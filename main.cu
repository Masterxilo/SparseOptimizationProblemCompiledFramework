#include "hip/hip_runtime.h"
/*
This program solves least-squares problems with energies of the form

    \sum_{P \in Q} \sum_{p \in P} ||f(select_p(x))||_2^2

Q gives a partitioning of the domain. In the simplest case, there is only one partition.

The solution to this may or may not be close to the solution to

    \sum_{p \in \Cup Q} ||f(select_p(x))||_2^2

*/

/*
SOPCompiled WL-library, c++ side.

Note: superceded by the SOPCompiled c++ interface/implementation.
TODO use that from here.

Compile with nvcc, after building the generated files with frameworkBuild.nb
(only needs to rerun when the external interface changes)

To make this CUDA debuggable, be sure to adjust then environment:

NSIGHT_CUDA_DEBUGGER=1

SetEnvironment["NSIGHT_CUDA_DEBUGGER" -> "1"]

Except for paul.h, standard/windows/cuda headers and generated files, this file is self-contained.
Generated required files:
    $CFormDefines
    f,df.cpp
    length(f)z.cpp
The Wolfram WSTP interface (WSTPMain.c, wstpExternC.cu, WSTPTemplateFile.tm) are optional.

TODO currently nvcc compilation seems to fail when df is present in the template file/wstp interface (linking error undefined reference to _Z*...df...). Just remove it from there.
The SOPCompiled interface only uses a handful of functions from here anyways (receive* and solve* basically)
*/

/*

Paul's CUDA/WSTP-Mathematica/C agnostic framework begins

FUNCTION declared functions can be called from Mathematica and CUDA whenever possible.

CPU_FUNCTION and CUDA_FUNCTION denote functions that use features only available on either platform (e.g. dues to CUDA API limitations or extensions)

Only primitive types can be passed back and forth automatically as of now.

*/

#include <paul.h>

#ifdef USE_WOLFRAM_LANGUAGE_INTERFACE

#define WL_WSTP_MAIN        // undefine to use main below to debug without mathematica 
#define WL_ALLOC_CONSOLE
#define WL_WSTP_PRE_MAIN
#include <paulwl.h> 

#endif












// todo put elsewhere, use CONSTANT() to be able to read them from outside -- this is currently not possible because interplay with the // preprocessor is not implemented in the WSTP wrapper code
/**/
CONSTANTD int lengthz =
#include "lengthz.cpp" /* generated for each problem */
;

CONSTANTD int lengthfz =
#include "lengthfz.cpp" /* generated for each problem */
;

#include "$WSTPWrappingCDefinesCUDA.h" /* generated for interface, used also for memory management & CUDA_CHECK_ERRORS*/

#ifdef USE_WOLFRAM_LANGUAGE_INTERFACE

#ifdef WL_WSTP_MAIN
#ifdef __HIPCC__
#include "wstpExternC.cu"              /* generated for interface */
#else
#include "wstpExternC.c"
#endif
#endif

#endif

#include "$CFormDefines.cpp"  /* generated for problem, rarely changes */  // Required for including *working* definitions of f and df -- this defines what times(x,y) etc. mean
#define x(i) input[i] /* definitions of f/df use x(i) to refer to input[], c.f. RIFunctionCForm* */

// TODO support these (_In_reads_(lengthz)) kinds of constant-sized vectors in the WSTP wrapper code
// TODO the wstp wrapper code should not be able to produce a caller for this because it does not know the length of input
// should not accept just anything
FUNCTION(void, f, (
    _In_reads_(lengthz) const float* const input,
    _Out_writes_all_(lengthfz) 
    float* const out
    ), "the local energy fvector computing function") {
#include "f.cpp" /* generated for each problem, depends on $CFormDefines*/
}

FUNCTION(void, df, (
    _In_range_(0, lengthz - 1) unsigned int const i,
    _In_reads_(lengthz) float const * const input,
    _Out_writes_all_(lengthfz)   
    float * const out
    ), "the derivatives along the i-th variable of the local energy fvector computing function") {
#include "df.cpp" /* generated for each problem, depends on $CFormDefines */
}
#undef x











// ---


// logging/debugging

GLOBAL(
    int,
    dprintEnabled,
    true,
    "if true, dprintf writes to stdout, otherwise dprintf does nothing"
    "It would be more efficient to compile with dprintf defined to nothing of course"
    "Default: true"
    );

#ifdef __CUDA_ARCH__
#define dprintf(formatstr, ...) {if (dprintEnabled) printf("CUDA " formatstr, __VA_ARGS__);}
#else
#define dprintf(formatstr, ...) {if (dprintEnabled) printf(formatstr, __VA_ARGS__);}
#endif

FUNCTION(void,
    print,
    (_In_z_ const char* const x),
    "prints a string to stdout"){
    printf("print: %s\n", x);
}


FUNCTION(
    void,
    printJ,
    (csparse* J),
    "prints a sparse matrix"
    ){
    if (dprintEnabled) cs_print(J);
}







#define copy mallocmemcpy

FUNCTION(int, lengthzGet, (), "lengthz"){ return lengthz; }
FUNCTION(int, lengthfzGet, (), "lengthfz") { return lengthfz; }



// SparseOptimizationProblem library

// --- Memory pool passed to the csparse library ---

// this is ideally some __shared__ memory in CUDA: In CUDA (I think) 
// C-style "stack" memory is first register based but then spills to main memory
// (is shared memory also used for the registers? Just another way to access the register file?)
// this memory does not need to be manually freed

// DEBUG TODO moved memory to global space for debugging -- move to __shared__ again.
// down the stack, no two functions should be calling SOMEMEM at the same time!

//__managed__ char memory[40000/*"Maximum Shared Memory Per Block" -> 49152*/ * 1000]; // TODO could allocate 8 byte sized type, should be aligned then (?)
//__managed__ bool claimedMemory = false; // makes sure that SOMEMEM is only called by one function on the stack

// "A default heap of eight megabytes is allocated if any program uses malloc() without explicitly specifying the heap size." -- want more 

void preWsMain() { // using a constructor to do this seems not to work
	runTests();
#if __HIPCC__
    int const mb = 400;
    printf("setting cuda malloc heap size to %d mb\n", mb);
    hipDeviceSetLimit(hipLimitMallocHeapSize, mb * 1000 * 1000); // basically the only memory we will use, so have some!
    CUDA_CHECK_ERRORS();
#endif
}
// TODO easily exceeded with lots of partitions on big scenes - small partitions don't need that much memory

// --- end of memory pool stuff ---


// one separate SOP (for one P in Q), shares only "x" with the global problem
// has custom y, p and values derived from that
// pointers are to __managed__ memory
// F() is another function for each partition P. It is defined as (f(s_p(x)))_{p in P}
struct SOPPartition {
    float* minusFx; size_t lengthFx; // "-F(x)"
    float* h; size_t lengthY; // "h, the update to y (subset of x, the parameters currently optimized over)"

    /*
    "amount of 'points' at which the function f is evaluated."
    "lengthP * lengthz is the length of xIndices, "
    "and sparseDerivativeZtoYIndices contains lengthP sequences of the form (k [k many z indices] [k many y indices]) "
    */
    size_t lengthP;

    // integer matrix of dimensions lengthz x lengthP, indexing into x to find the values to pass to f
    int* xIndices;

    // Used to construct J, c.f. SOPJF
    int* sparseDerivativeZtoYIndices; // serialized form of this ragged array

    /*
    "the indices into x that indicate where the y are"
    "needed to write out the final update h to the parameters"
    */
    int* yIndices; /* lengthY */
};

GLOBALDYNAMICARRAY(
    SOPPartition, partitionTable, partitions,
    "partitions of the SOPD, allocated on call to setPartitions");

FUNCTION(void, writeJFx, (csparse* const J, const size_t i, const size_t j, const float x),
    "set J(i, j) = x"
    ) {
    assert(J);
    assert(cs_is_triplet(J));
    assert(i < J->m && j < J->n);
    assert(J->nz + 1 <= J->nzmax); // matrix should not become overful
    assertFinite(x);

    cs_entry(J, i, j, x);
}

FUNCTION(void, writeFx, (SOPPartition* const sop, const size_t i, const float val), "F(x)_i = val") {
    assert(i < sop->lengthFx);
    assert(sop->minusFx);
    assertFinite(val);

    sop->minusFx[i] = -val;
}

GLOBALDYNAMICARRAY(
    float, x, lengthx,
    "stores the current data fvector 'x' which is updated to reduce the energy ||F(x)||^2"
    );



// -----------------------
/*
Given access to :

int lengthP
int lengthY
const int lengthz (compile-time constant)
const int lengthfz (compile-time constant)
f(fz_out, z)
df(i, fz_out, z)
float* x
int* xIndices (a list of indices into x, lengthfz * n many)
int* sparseDerivativeZtoYIndices (a list of n lists of integers of the structure {k   (deriveby - k integers from 0 to argcount(f)-1) (store at - k integers from 0 to y_length-1)

This creates the fvector
Fx
and the sparse matrix
JFx

By calling

void writeFx(int i, float val)
void writeJFx(int i, int j, float val)

using only elementary C constructs
*/
// TODO move these functions to SOPPartition instead of passing the pointer all the time
FUNCTION(void, readZ, (
    SOPPartition* const sop,
    _Out_writes_all_(lengthz) float* z,
    const size_t rowz
    ), "z = x[[xIndices[[rowz;;rowz+lengthz-1]]]]"){
    assert(divisible(rowz, lengthz));

    extract_by_indices(z, x, lengthx, sop->xIndices + rowz, lengthz); // z = x[[xIndices]]
}

FUNCTION(void, readZandSetFxRow, (
    SOPPartition* const sop,
    _Out_writes_all_(lengthz) float* z,
    const size_t rowz,
    const size_t rowfz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]] = f(z) and return the z = x[[xIndices[[rowz;;rowz+lengthz-1]]]] required for that"){
    assert(divisible(rowz, lengthz));
    assert(divisible(rowfz, lengthfz));

    readZ(sop, z, rowz); // z = x[[xIndices]]

    float fz[lengthfz];
    f(z, fz); // fz = f(z)

    DO(i, lengthfz) writeFx(sop, rowfz + i, fz[i]); // Fx[[rowfz;;rowfz+lengthfz-1]] = fz
}

FUNCTION(void, setFxRow, (
    SOPPartition* const sop,
    const size_t rowz,
    const size_t rowfz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]]"){
    float z[lengthz];
    readZandSetFxRow(sop, z, rowz, rowfz);
}

FUNCTION(void, buildFx, (SOPPartition* const sop), "from the current x, computes just F(x)"){
    size_t rowz = 0;
    size_t rowfz = 0;

    FOR(size_t, i, 0, sop->lengthP, (rowz += lengthz, rowfz += lengthfz, 1)) MAKE_CONST(rowz) MAKE_CONST(rowfz) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        setFxRow(sop, rowz, rowfz);
    }
}

FUNCTION(void, buildFxandJFx, (SOPPartition* const sop, csparse* const J, bool buildFx),
    "from the current x, computes F(x) [if buildFx == true] and JF(x)"
    "Note that J is stored into the matrix pointed to"
    "this J must by in triplet form and have allocated enough space to fill in the computed df"
    ) {
    assert(cs_is_triplet(J));
    auto* currentSparseDerivativeZtoYIndices = sop->sparseDerivativeZtoYIndices;
    size_t rowz = 0;
    size_t rowfz = 0;

    FOR(size_t, i, 0, sop->lengthP, (rowz += lengthz, rowfz += lengthfz, 1)) MAKE_CONST(rowz) MAKE_CONST(rowfz) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        float z[lengthz];
        if (buildFx)
            readZandSetFxRow(sop, z, rowz, rowfz);
        else
            readZ(sop, z, rowz);

        // deserialize sparseDerivativeZtoYIndices, c.f. flattenSparseDerivativeZtoYIndices
        // convert back to two lists of integers of the same length (K)
        const int K = *currentSparseDerivativeZtoYIndices++;
        assert(K <= lengthz);
        const int* const zIndices = currentSparseDerivativeZtoYIndices; currentSparseDerivativeZtoYIndices += K;
        const int* const yIndices = currentSparseDerivativeZtoYIndices; currentSparseDerivativeZtoYIndices += K;

        // construct & insert localJ columnwise
        DO(k, K) {
            const int zIndex = zIndices[k];
            const int yIndex = yIndices[k];

            assert(zIndex < lengthz);
            assert(yIndex < sop->lengthY);

            float localJColumn[lengthfz];
            df(zIndex, z, localJColumn);

            // put in the right place (starting at rowfz, column yIndex)
            DO(j, lengthfz) {
                writeJFx(J, rowfz + j, yIndex, localJColumn[j]);
            }
        }
    }
}
// -----------------------

// Core algorithms

FUNCTION(void,
    solve,
    (SOPPartition* const sop, csparse const * const J, _In_ AllocationAligned8State mem),
    "assumes x, -Fx and J have been built"
    "computes the adjustment fvector h, which is the least-squares solution to the system"
    "Jh = -Fx"
    ""
    "Allocates temporaries in mem"
    ) {
    assert(J && x && sop && sop->minusFx && sop->h);
    assert(cs_is_compressed_col(J));

    printf("sparse leastSquares (cg) %d x %d... (this might take a while)\n",
        J->m, J->n);

    assert(sop->lengthY > 0);

    // h must be initialized -- initial guess -- use 0
    memset(sop->h, 0, sizeof(float) * sop->lengthY); // not lengthFx! -- in page writing error -- use struct fvector to keep fvector always with its length (existing solutions?)

    cs_cg(J, sop->minusFx, sop->h, mem);

    dprintf("h:\n"); printv(sop->h, sop->lengthY);
    assertFinite(sop->h, (unsigned int)sop->lengthY);
}

FUNCTION(
    float,
    norm2Fx,
    (SOPPartition const * const sop), "Assuming F(x) is computed, returns ||F(x)||_2^2"
    ) {
    assert(sop->minusFx);
    float x = 0;
    DO(i, sop->lengthFx) x += sop->minusFx[i] * sop->minusFx[i];
    return assertFinite(x);
}

FUNCTION(
    float,
    addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore,
    (SOPPartition * const sop),
    "scales h such that F(x0 + h) < F(x) in the 2-norm and updates x = x0 + h"
    "returns total energy delta achieved which should be negative but might not be when the iteration count is exceeded"
    ) {
    assert(sop);
    assert(sop->yIndices);
    assert(sop->minusFx);
    assert(x);

    // determine old norm
    const float norm2Fatx0 = norm2Fx(sop);
    dprintf("||F[x0]||_2^2 = %f\n", norm2Fatx0);

    // add full h
    float lambda = 1.;
    dprintf("x = "); printv(x, lengthx);
    axpyWithReindexing(x, lengthx, lambda, sop->h, (unsigned int*)sop->yIndices, sop->lengthY); // xv = x0 + h
    dprintf("x = x0 + h = "); printv(x, lengthx); 

    buildFx(sop);
    float norm2Faty0 = norm2Fx(sop);
    dprintf("||F[x0 + h]||_2^2 = %f\n", norm2Faty0);


    // Reduce step-size if chosen step does not lead to reduction by subtracting lambda * h
    size_t n = 0; // safety net, limit iterations
    while (norm2Faty0 > norm2Fatx0 && n++ < 20) {
        lambda /= 2.;
        axpyWithReindexing(x, lengthx, -lambda, sop->h, (unsigned int*)sop->yIndices, sop->lengthY); // xv -= lambda * h // note the -!

        dprintf("x = "); printv(x, lengthx); 

        buildFx(sop); // rebuild Fx after this change to x
        norm2Faty0 = norm2Fx(sop); // reevaluate norm
        dprintf("reduced stepsize, lambda =  %f, ||F[y0]||_2^2 = %f\n", lambda, norm2Faty0);
    }
    dprintf("optimization finishes, total energy change: %f\n", norm2Faty0-norm2Fatx0);
    /*assert(norm2Faty0 - norm2Fatx0 <= 0.);*/ // might not be true if early out was used
    return norm2Faty0 - norm2Fatx0;
}

// Interface

FUNCTION(
    void,
    getY,
    (
    int partition,
    _Out_writes_all_(lengthY) float* const outY,
    int lengthY
    ),
    "Extracts from x the elements that are y"
    "lengthY must match the length passed at initialization"
    "TODO make WSTP support having global variables specify the necessary array length, support non size_t lengths"
    "then we wouldn't need to supply the redundant lengthY here"
    )
{
    assert(partition >= 0 && partition < partitions);
    const SOPPartition* const sop = &partitionTable[partition];
    assert(x && sop && sop->yIndices && lengthx && sop->lengthY);
    assert(sop->lengthY <= lengthx);
    assert(lengthY == sop->lengthY);
	extract_by_indices(outY, ::x
		, (int)::lengthx, (int*)sop->yIndices, (int)sop->lengthY // TODO should be unsigned
		);
}

// TODO size_t not properly supported because of WSTP receiving only Integer64
CPU_FUNCTION(
    void, setPartitions, (size_t newPartitionsCount), "set the amount of partitions") {

    // free old stuff first
    assert(partitions >= 0);
    DO(i, partitions) {
        SOPPartition* const sop = &partitionTable[i];

        memoryFree(sop->sparseDerivativeZtoYIndices);
        memoryFree(sop->xIndices);
        memoryFree(sop->yIndices);
        memoryFree(sop->minusFx);
        memoryFree(sop->h);
    }
    memoryFree(partitionTable);

    // allocate
    ::partitions = newPartitionsCount;
    partitionTable = tmalloczeroed<SOPPartition>(partitions); // pointers not yet initialized
}

CPU_FUNCTION(
    void,
    receiveSharedOptimizationData,
    (
    _In_reads_(xLength) const float* const xI, const size_t xLength
    ),
    "Receives x"
    ) {
    memoryFree(x);
    ::x = copy(xI, xLength);
    ::lengthx = xLength;
}

// macro for indexing into partitionTable, sop = partitionTable[partition]
#define extractSop(partition) assert(partition >= 0 && partition < partitions); SOPPartition* const sop = &partitionTable[partition];

CPU_FUNCTION(
    void,
    receiveOptimizationData,
    (
    const int partition,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndicesI, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndicesI, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndicesI, const size_t yIndicesLength
    ),
    "Receives sparseDerivativeZtoYIndices, xIndices and yIndices"
    "Appropriately sized vectors for receiving these data items are newly allocated in __managed__ memory, hence this is a CPU only function"
    ) {
    extractSop(partition);

    sop->sparseDerivativeZtoYIndices = copy(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength);
    sop->xIndices = copy(xIndicesI, xIndicesLength);
    sop->yIndices = copy(yIndicesI, yIndicesLength);

    assert(lengthz > 0);
    assert(divisible(xIndicesLength, lengthz));
    assert(lengthfz > 0);

    sop->lengthP = xIndicesLength / lengthz;
    sop->lengthY = yIndicesLength;
    sop->lengthFx = lengthfz * sop->lengthP;

    sop->minusFx = tmalloc<float>(sop->lengthFx);

    sop->h = tmalloc<float>(sop->lengthY);
}

FUNCTION(
    void,
    buildFxAndJFxAndSolve,
    (SOPPartition * const sop, bool buildFx),
    "using current data, builds JFx (and Fx) and solves the least squares problem"
    "optionally does not compute Fx, assuming it is current with the x data (true after every solve)"
    ""
    "Note that we must do the solving right here, because this function handles the memory needed by J"
    "the solution is then accessible in h for further processing (updating x at yIndices)"
    ""
    "sop is passed here, not partition. Use buildFxAndJFxAndSolveRepeatedly as the external interface"
    )
{
    // Build F and JF
    const size_t maxNNZ = (lengthfz*lengthz) * sop->lengthP; // very pessimistic estimate/overestimation: assume every derivative figures for every P -- usually not all of them will be needed
    // ^^ e.g. in vsfs the 3 color channels are all not optimized over, neither doriginal

    // consider using dynamic allocation in SOMEMEM!

	aa8_malloc_raii mem(8 * 1000 * 1000);
    dprintf("allocating sparse matrix for %d entries\n", maxNNZ);
    csparse* J = cs_spalloc(sop->lengthFx, sop->lengthY, maxNNZ, 1, mem.aa8_state); // might run out of memory here

    dprintf("buildFxandJFx\n");
    buildFxandJFx(sop, J, buildFx);

    dprintf("used %d of %d allocated spaces in J\n", J->nz, J->nzmax);
    assert(J->nz > 0); // there must be at least one (nonzero) entry in the jacobian, otherwise we have taken the derivative only over variables no ocurring (or no variables at all!)

    J = cs_triplet(J, mem.aa8_state); // "optimizes storage of J, after which it may no longer be modified" 
    // TODO recycle memory

    // State
    dprintf("-F(x):\n");
    printv(sop->minusFx, sop->lengthFx);
    dprintf("JF(x):\n");
    printJ(J);

    // Solve
    dprintf("solve:\n");
    solve(sop, J, mem.aa8_state); // TODO allocates even more memory
}

FUNCTION(
    void,
    buildFxAndJFxAndSolveRepeatedly,
    (const int partition, const int iterations),
    "using current data, builds JFx (and Fx) and solves the least squares problem"
    "then does a gradient descent step"
    "reapeats this whole process as often as desired"
    )
{ 
    extractSop(partition);

    // TODO we might want to do this externally
    printf("\n=== buildFxAndJFxAndSolveRepeatedly %d times in partition %d of %d ===\n", iterations, partition, partitions);
    assert(iterations > 0); // TODO iterations should be size_t
    
    DO(i, iterations) {
        bool buildFx = i == 0; // Fx is always up-to date after first iteration

        buildFxAndJFxAndSolve(sop, buildFx);
        const float delta = addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore(sop);
        if (delta > -0.001) {
            dprintf("delta was only %f, stopping optimization\n", delta);
            return;
        }
    }
}

FUNCTION(
    void,
    buildFxAndJFxAndSolveRepeatedlyThreadIdPartition,
    (const int iterations),
    "buildFxAndJFxAndSolveRepeatedly on the partition given by linear_global_threadId."
    "does nothing when linear_global_threadId is >= partitions"
    ""
    "TODO this should be the block id, threads in the same block should cooperate in the same partition"
    )
{
    if (linear_global_threadId() >= partitions) {
        dprintf("\n--- thread id %d has nothing to do  - there are only %d partitions\n", linear_global_threadId(), partitions);
        return;
    }

    printf("\n=== Starting work on partition %d in the thread of the same id ===\n", linear_global_threadId());
    buildFxAndJFxAndSolveRepeatedly(linear_global_threadId(), iterations);
}

// Prototyping functions

FUNCTION(void,
    receiveAndPrintOptimizationData,
    (
    const int lengthz,
    const int lengthfz,

    _In_reads_(xLength) const float* const x, const size_t xLength,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndices, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndices, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndices, const size_t yIndicesLength
    ),
    "Receives x, sparseDerivativeZtoYIndices, xIndices and yIndices, checks and prints them,"
    "emulating arbitrary lengthz, lengthfz"
    "Note: lengthz, lengthfz are fixed at compile-time for other functions"
    "This is a prototyping function that does not allocate or copy anything"
    "use for testing"
    ) {

    const int lengthP = xIndicesLength / lengthz;
    const int lengthY = yIndicesLength;
    const int lengthFx = lengthfz * lengthP;
    const int maxNNZ = (lengthfz*lengthz) * lengthP; // could go down from lengthz to maximum k in sparseDerivativeZtoYIndices
     // or just the actual sum of all such k

    dprintf("lengthz: %d\n", lengthz);
    dprintf("lengthfz: %d\n", lengthfz);
    dprintf("lengthP: %d\n", lengthP);
    dprintf("lengthY: %d\n", lengthY);
    dprintf("lengthFx: %d\n", lengthFx);
    dprintf("maxNNZ: %d\n", maxNNZ);

    assert(lengthz > 0);
    assert(lengthfz > 0);
    assert(lengthY > 0);

    dprintf("x:\n");
    printv(x, xLength);

    dprintf("sparseDerivativeZtoYIndices:\n");
    const int* p = sparseDerivativeZtoYIndices;
    DO(i, lengthP) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        int k = *p++; 
        assert(k <= lengthz);
        dprintf("---\n");
        printd(p, k); p += k;
        dprintf("-->\n");
        printd(p, k); p += k;
        dprintf("---\n");
    }
    assert(p == sparseDerivativeZtoYIndices + sparseDerivativeZtoYIndicesLength);

    dprintf("xIndices:\n");
    p = xIndices;
    DO(i, lengthP) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        printd(p, lengthz);
        p += lengthz;
    }
    assert(p == xIndices + xIndicesLength);
    assertEachInRange(xIndices, xIndicesLength, 0, (int)xLength - 1);

    dprintf("yIndices:\n");
    printd(yIndices, yIndicesLength);
    assertEachInRange(yIndices, yIndicesLength, 0, (int)xLength - 1);
}



FUNCTION(
    void,
    makeAndPrintSparseMatrix,
    (
    _In_ const size_t m,
    _In_ const size_t n,
    _In_reads_(xlen) float* x,
    size_t xlen,

    _In_reads_(ijlen) int* ij,
    const size_t ijlen
    ),
    "Creates a sparse matrix from a list of values and a list of pairs of (i, j) indices specifying where to put the corresponding values (triplet form) then cs_prints it to stdout both in column form and vanilla."
	""
	"The matrix should be fairly small because the required intermediate storage is on the stack."
    "Note: This is a prototyping function without any further purpose"
    ) {
    assert(2 * xlen == ijlen);
    assert(xlen <= m*n); // don't allow repeated entries

	char _mem[1000 * 10];
	auto mem = aa8_make(_mem);
    csparse* const A = cs_spalloc(m, n, xlen, 1, mem);

    while (xlen--) {
        int i = *ij++;
        int j = *ij++;
        cs_entry(A, i, j, *x++);
    }

    cs_print(A);

    printf("compress and print again:\n");
    const csparse* const B = cs_triplet(A, mem);
    cs_print(B);
    printf("done--\n");
}

TEST(makeAndPrintSparseMatrix1) {
	unsigned int const count = 1;
	float x[] = { 1. };
	int ij[] = { 0, 0 };
	makeAndPrintSparseMatrix(1, 1, x, count, ij, 2 * count);
}







// Misc


FUNCTION(void, testMain, (), "collection of some tests") {
    dprintf("-- sparse optimize solver --\n");
    dprintf("Compiled with:\n");
    dprintf("lengthz: %d\n", lengthz);
    dprintf("lengthfz: %d\n", lengthfz);

    dprintf("POST begins:\n");

    // expect i: 0-9
    FOR(int,i,0,10,1)
        dprintf("i: %d\n", i);

	int i = 0;
	REPEAT(10)
		dprintf("rep i: %d\n", i++);
}



GLOBAL(int, xx, 0, "xx");
GLOBAL(float, a, 0, "a");
GLOBAL(float, b, 0, "b");
GLOBAL(float, yy, 0, "yy");

FUNCTION(void, xx43, (), "sets xx to 43") {
    xx = 43;
}

FUNCTION(int, get42, (), "returns the answer to the universe life and everything") {
    return 42;
}

FUNCTION(int, add, (int x, int y), "x + y") {
    return x + y;
}

FUNCTION(float, addf, (float x, float y), "x + y with floats") {
    return x + y;
}

FUNCTION(int, multiout, (_Inout_updates_(l) int* x, int l), "returns more than one thing: when called via WSTP this will return an Association with all results") {
    *x = 0;
    return 1;
}



// this can be compiled as a program rather as well as a Wolfram Language LinkedLibrary
#ifndef WL_WSTP_MAIN 
int main() {
    //mainc<<<1,1>>>();
    //hipDeviceSynchronize();
#if __HIPCC__
    auto e = 74; printf("hipGetLastError %d %s %s\n", e, hipGetErrorName((hipError_t)e), hipGetErrorString((hipError_t)e));
#endif
    return 0;
}
#endif
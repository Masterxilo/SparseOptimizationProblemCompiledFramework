#include "hip/hip_runtime.h"
/*
To make this debuggable, be sure to adjust then environment:
NSIGHT_CUDA_DEBUGGER=1
SetEnvironment["NSIGHT_CUDA_DEBUGGER" -> "1"]
*/

// WSTP interfacing
#define WL_WSTP_MAIN // undefine to use main below to debug without mathematica
#define WL_ALLOC_CONSOLE
#include "framework.h"

#include "wstpExternC.cu"

#include "csparseminimal.cpp"

#define copy mallocmemcpy


FUNCTION(int, lengthzGet, (), "lengthz"){ return lengthz; }
FUNCTION(int, lengthfzGet, (), "lengthfz") { return lengthfz; }

// logging/debugging

GLOBAL(
    int,
    dprintEnabled,
    true,
    "if true, dprintf writes to stdout, otherwise dprintf does nothing"
    "It would be more efficient to compile with dprintf defined to nothing of course"
    );

#ifdef __CUDA_ARCH__
#define dprintf(formatstr, ...) {if (dprintEnabled) printf("CUDA " formatstr, __VA_ARGS__);}
#else
#define dprintf(formatstr, ...) {if (dprintEnabled) printf(formatstr, __VA_ARGS__);}
#endif

FUNCTION(void,
    print,
    (_In_z_ const char* const x),
    "prints a string to stdout"){
    printf("print: %s\n", x);
}

/* size_t n not const because the implementation modifies it
changes not visible outside anyways:
the last const spec is always an implementation detail, but can indicate conceptual thinking */
FUNCTION(
    void,
    printv,
    (_In_reads_(n) const real* v, size_t n),
    "dprints a vector of doubles, space separated and newline terminated"
    ) {
    while (n--) dprintf("%f ", *v++); dprintf("\n");
}

FUNCTION(
    void,
    printd,
    (_In_reads_(n) const int* v, size_t n),
    "dprints a vector of integers, space separated and newline terminated"
    ) {
    while (n--) dprintf("%d ", *v++); dprintf("\n");
}

FUNCTION(
    void,
    printJ,
    (cs* J),
    "prints the matrix J, (i.e. dF/dy (x)). Cannot be used externally."
    ){
    if (dprintEnabled) cs_print(J);
}


// pure functions, utilities

FUNCTION(
    void,
    assertEachInRange,
    (
    _In_reads_(len) const int* v, 
    size_t len, 
    const int min,
    const int max
    ),
    "computes the same as BoolEval[min <= v <= max]"
    ) {
    while (len--) {
        assert(min <= *v && *v <= max);
        v++;
    }
}

FUNCTION(
    void,

    axpyWithReindexing,

    (
    _Inout_updates_(targetLength) real* const targetBase,
    const size_t targetLength,
    real const a,

    _In_reads_(targetIndicesAndAddedValuesLength) const real* const addedValues,
    _In_reads_(targetIndicesAndAddedValuesLength) const int* const targetIndices,
    const size_t targetIndicesAndAddedValuesLength
    ),
    "targetBase[[targetIndices]] += a * addedValues. Repeated indices are not supported, so addedValues cannot be longer than the target."
    "Note that not necessarily all of target is updated"

    ) {
    assertFinite(a);
    assert(targetIndicesAndAddedValuesLength <= targetLength);
    dprintf("axpyWithReindexing %f %d %d\n", a, targetLength, targetIndicesAndAddedValuesLength);

    dprintf("target before:\n"); printv(targetBase, targetLength);
    dprintf("targetIndices:\n"); printd(targetIndices, targetIndicesAndAddedValuesLength);
    dprintf("addedValues:\n"); printv(addedValues, targetIndicesAndAddedValuesLength);
    
    assertEachInRange(targetIndices, targetIndicesAndAddedValuesLength, 0, targetLength - 1);

    DO(j, targetIndicesAndAddedValuesLength)
        assertFinite(targetBase[targetIndices[j]] += addedValues[j] * a);

    dprintf("target after:\n"); printv(targetBase, targetLength);
}

FUNCTION(void, extract, (
    _Out_writes_all_ (sourceIndicesAndTargetLength) real* const target,

    _In_reads_(sourceLength) const real* const source,
    const size_t sourceLength,

    _In_reads_(sourceIndicesAndTargetLength) const int* const sourceIndices,
    const size_t sourceIndicesAndTargetLength
    ),
    "target = source[[sourceIndices]]. Note that all of target is updated"
    ) {
    assertEachInRange(sourceIndices, sourceIndicesAndTargetLength, 0, sourceLength - 1);

    DO(i, sourceIndicesAndTargetLength)
        target[i] = source[sourceIndices[i]];
}

// SparseOptimizationProblem library

// --- Memory pool passed to the csparse library ---

// this is ideally some __shared__ memory in CUDA: C-style "stack" memory is first register based but then spills to main memory
// (is shared memory also used for the registers? Just another way to access the register file?)
// this memory does not need to be manually freed

// DEBUG TODO moved memory to global space for debugging -- move to __shared__ again.
// down the stack, no two functions should be calling SOMEMEM at the same time!

//__managed__ char memory[40000/*"Maximum Shared Memory Per Block" -> 49152*/ * 1000]; // TODO could allocate 8 byte sized type, should be aligned then (?)
//__managed__ bool claimedMemory = false;

#define SOMEMEM() \
    char memory[40000/*"Maximum Shared Memory Per Block" -> 49152*/ ];\
    char* mem = (char*)(((unsigned long long)memory+7) & (~ 0x7ull)); /* align on 8 byte boundary */\
    assert(aligned(mem, 8) && after(mem, memory));\
    int memsz = sizeof(memory)-8;/*be safe*/ \
    //assert(!claimedMemory); claimedMemory = true;

#define FREESOMEMEM() //claimedMemory = false;

#define SOMEMEMP mem, memsz

// --- ---

FUNCTION(void, writeJFx, (cs* const J, const size_t i, const size_t j, const real x),
    "set J(i, j) = x"
    ) {
    assert(J);
    assert(cs_is_triplet(J));
    assert(i < J->m && j < J->n);
    assert(J->nz + 1 <= J->nzmax);
    assertFinite(x);

    cs_entry(J, i, j, x);
}

FUNCTION(void, writeFx, (const size_t i, const real val), "F(x)_i = val") {
    assert(i < lengthFx);
    assert(minusFx);
    assertFinite(val);

    minusFx[i] = -val;
}

GLOBALDYNAMICARRAY(
    real, x, lengthx,
    "stores the current data vector 'x' which is updated to reduce the energy ||F(x)||^2"
    );

// Required for including definitions of f and df:
#include "$CFormDefines.cpp"
#define x(i) input[i]

// TODO generate them like that (FUNCTION), add parameters of length for in and out that are asserted
// but since its and output parameter it still cannot be called from WL --> 
//  make special declarations for such variables (microsoft has a standard annotation)

FUNCTION(void, f, (
    _In_reads_(lengthz) const real* const input, // TODO support these kinds of constant-sized vectors in the WSTP wrapper code
    _Out_writes_(lengthfz) real* const out), "the local energy vector computing function") {
#include "f.cpp" // generated
}
FUNCTION(void, df, (int const i,
    _In_reads_(lengthz) real const * const input,
    _Out_writes_(lengthfz) real * const out
    ), "the derivatives along the i-th variable of the local energy vector computing function") {
#include "df.cpp" // generated
}
#undef x

GLOBALDYNAMICARRAY(
    real, minusFx, lengthFx,
    "-F(x)"
    );

GLOBALDYNAMICARRAY(
    real, h, lengthY,
    "h, the update to y, subset of x, the parameters currently optimized over"
    );

// Could be const on GPU side

GLOBAL(
    int, lengthP, 0,
    "amount of 'points' at which the function f is evaluated."
    "lengthP * lengthz is the length of xIndices, "
    "and sparseDerivativeZtoYIndices contains lengthP sequences of the form (k [k many] [k many]) "
    );

// Could be const on GPU side
// integer matrix of dimensions lengthz x lengthP, indexing into x to find the values to pass to f
__managed__ int* xIndices = 0;

// Could be const on GPU side
// Used to construct J, c.f. SOPJF
__managed__ int* sparseDerivativeZtoYIndices = 0;

// Could be const on GPU side
GLOBALDYNAMICARRAY_SHAREDLENGTH(
    int, yIndices, lengthY,
    "the indices into x that indicate where the y are"
    );


// -----------------------
/*
Given access to (somehow):

int lengthP
int lengthY
const int lengthz (compile-time constant)
const int lengthfz (compile-time constant)
f(fz_out, z)
df(i, fz_out, z)
real* x
int* xIndices (a list of indices into x, lengthfz * n many)
int* sparseDerivativeZtoYIndices (a list of n lists of integers of the structure {k   (deriveby - k integers from 0 to argcount(f)-1) (store at - k integers from 0 to y_length-1)

This creates the vector
Fx
and the sparse matrix
JFx

By calling

void writeFx(int i, real val)
void writeJFx(int i, int j, real val)

using only elementary C constructs
*/

FUNCTION(void, readZ, (
    _Out_writes_all_(lengthz) real* z,
    const size_t rowz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]] = f(z) and return the z required for that"){
    assert(divisible(rowz, lengthz));

    extract(z, x, lengthx, xIndices + rowz, lengthz); // z = x[[xIndices]]
}

FUNCTION(void, readZandSetFxRow, (
    _Out_writes_all_(lengthz) real* z,
    const size_t rowz, 
    const size_t rowfz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]] = f(z) and return the z required for that"){
    assert(divisible(rowz, lengthz));
    assert(divisible(rowfz, lengthfz));

    readZ(z, rowz); // z = x[[xIndices]]

    real fz[lengthfz];
    f(z, fz); // fz = f(z)

    DO(i, lengthfz) writeFx(rowfz + i, fz[i]); // Fx[[rowfz;;rowfz+lengthfz-1]] = fz
}

FUNCTION(void, setFxRow, (
    const size_t rowz,
    const size_t rowfz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]]"){
    real z[lengthz];
    readZandSetFxRow(z, rowz, rowfz);
}

FUNCTION(void, buildFx,(), "from the current x, computes F(x)" ){
    size_t rowz = 0;
    size_t rowfz = 0;

    FOR(size_t, i, 0, lengthP, (rowz += lengthz, rowfz += lengthfz, 1)) MAKE_CONST(rowz) MAKE_CONST(rowfz) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        setFxRow(rowz, rowfz);
    }
}

FUNCTION(void, buildFxandJFx, (cs* const J, bool buildFx), 
    "from the current x, computes F(x) and JF(x)"
    "note that J is stored into the matrix pointed to"
    "this J must have the right size is stored somewhere, in CUDA idally in shared memory"
    ) {
    auto* currentSparseDerivativeZtoYIndices = sparseDerivativeZtoYIndices;
    size_t rowz = 0;
    size_t rowfz = 0;

    FOR(size_t, i, 0, lengthP, (rowz += lengthz, rowfz += lengthfz, 1)) MAKE_CONST(rowz) MAKE_CONST(rowfz) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        real z[lengthz];
        if (buildFx)
            readZandSetFxRow(z, rowz, rowfz);
        else
            readZ(z, rowz);

        // deserialize sparseDerivativeZtoYIndices, c.f. flattenSparseDerivativeZtoYIndices
        // convert back to two lists of integers of the same length (K)
        const int K = *currentSparseDerivativeZtoYIndices++;
        assert(K <= lengthz);
        const int* const zIndices = currentSparseDerivativeZtoYIndices; currentSparseDerivativeZtoYIndices += K;
        const int* const yIndices = currentSparseDerivativeZtoYIndices; currentSparseDerivativeZtoYIndices += K;

        // construct & insert localJ columnwise
        DO(k, K) {
            const int zIndex = zIndices[k];
            const int yIndex = yIndices[k];

            assert(zIndex < lengthz);
            assert(yIndex < lengthY);

            real localJColumn[lengthfz];
            df(zIndex, z, localJColumn);

            // put in the right place (starting at rowfz, column yIndex)
            DO(j, lengthfz) {
                writeJFx(J, rowfz + j, yIndex, localJColumn[j]);
            }
        }
    }
}
// -----------------------

// Core algorithms

FUNCTION(void,
    solve,
    (cs const * const J, MEMPOOL),
    "assumes x, -Fx and J have been built"
    "computes the adjustment vector h, which is the least-squares solution to the system"
    "Jh = -Fx"
    ) {
    assert(J && x && minusFx && h);
    assert(cs_is_compressed_col(J));

    dprintf("sparse leastSquares (cg) %d x %d... (this might take a while)\n",
        J->m, J->n);

    assert(lengthY > 0);

    // h must be initialized -- initial guess -- use 0
    memset(h, 0, sizeof(real) * lengthFx);

    cs_cg(J, minusFx, h, MEMPOOLPARAM);

    dprintf("h:\n"); printv(h, lengthY);

}

FUNCTION(
    real,
    norm2Fx,
    (), "Assuming F(x) is computed, returns ||F(x)||_2^2" 
    ) {
    assert(minusFx);
    real x = 0;
    DO(i, lengthFx) x += minusFx[i] * minusFx[i];
    return assertFinite(x);
}

FUNCTION(
    float,
    addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore,
    (),
    "scales h such that F(x0 + h) < F(x) in the 2-norm and updates x = x0 + h"
    "returns total energy delta achieved which should be negative but might not be when the iteration count is exceeded"
    ) {
    assert(yIndices);
    assert(x);

    // determine old norm
    const real norm2Fatx0 = norm2Fx();
    dprintf("||F[x0]||_2^2 = %f\n", norm2Fatx0);

    // add full h
    real lambda = 1.;
    dprintf("x = "); printv(x, lengthx);
    axpyWithReindexing(x, lengthx, lambda, h, yIndices, lengthY); // xv = x0 + h
    dprintf("x = x0 + h = "); printv(x, lengthx); 

    buildFx();
    real norm2Faty0 = norm2Fx();
    dprintf("||F[x0 + h]||_2^2 = %f\n", norm2Faty0);


    // Reduce step-size if chosen step does not lead to reduction by subtracting lambda * h
    size_t n = 0; // safety net, limit iterations
    while (norm2Faty0 > norm2Fatx0 && n++ < 20) {
        lambda /= 2.;
        axpyWithReindexing(x, lengthx, -lambda, h, yIndices, lengthY); // xv -= lambda * h // note the -!

        dprintf("x = "); printv(x, lengthx); 

        buildFx();
        norm2Faty0 = norm2Fx(); // reevaluate norm
        dprintf("reduced stepsize, lambda =  %f, ||F[y0]||_2^2 = %f\n", lambda, norm2Faty0);
    }
    dprintf("optimization finishes, total energy change: %f\n", norm2Faty0-norm2Fatx0);
    /*assert(norm2Faty0 - norm2Fatx0 <= 0.);*/
    return norm2Faty0 - norm2Fatx0;
}

// Interface

FUNCTION(
    void,
    getY,
    (
    _Out_writes_all_(lengthY) real* const outY,
    int lengthY
    ),
    "Extracts from x the elements that are y"
    "lengthY must match the length passed at initialization"
    "TODO make WSTP support having global variables specify the necessary array length, support non size_t lengths"
    )
{
    assert(lengthY == ::lengthY);
    extract(outY, ::x, ::lengthx, ::yIndices, lengthY);
}

CPU_FUNCTION(
    void,
    receiveOptimizationData,
    (
    _In_reads_(xLength) const real* const xI, const size_t xLength,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndicesI, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndicesI, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndicesI, const size_t yIndicesLength
    ),
    "Receives x, sparseDerivativeZtoYIndices, xIndices and yIndices"
    "Appropriately sized vectors for receiving these data items is allocated in __managed__ memory, hence this is a CPU only function"
    ) {
    // compiler seems confused when both are called x
    ::x = copy(xI, xLength); // TODO modifiable Lvalue problem
    ::sparseDerivativeZtoYIndices = copy(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength);
    ::xIndices = copy(xIndicesI, xIndicesLength);
    ::yIndices = copy(yIndicesI, yIndicesLength);

    ::lengthP = xIndicesLength / lengthz;
    ::lengthY = yIndicesLength;
    ::lengthFx = lengthfz * lengthP;
    ::lengthx = xLength;
    ::minusFx = tmalloc<real>(lengthFx);

    ::h = tmalloc<real>(lengthY);
}

FUNCTION(
    void,
    buildFxAndJFxAndSolve,
    (bool buildFx),
    "using current data, builds JFx (and Fx) and solves the least squares problem"
    "optionally does not compute Fx, assuming it is current with the x data (true after every solve)"
    ""
    "Note that we must do the solving right here, because this function handles the memory needed by J"
    "the solution is then accessible in h for further processing (updating x at yIndices)"
    )
{
    // Build F and JF
    const size_t maxNNZ = (lengthfz*lengthz) * lengthP;

    SOMEMEM();
    cs* J = cs_spalloc(lengthFx, lengthY, maxNNZ, 1, SOMEMEMP);

    buildFxandJFx(J, buildFx);

    J = cs_triplet(J, SOMEMEMP); // "optimizes storage of J, after which it may no longer be modified"

    dprintf("-F(x):\n");
    printv(minusFx, lengthFx);
    dprintf("JF(x):\n");
    printJ(J);

    dprintf("solve:\n");
    solve(J, SOMEMEMP);

    FREESOMEMEM();
}

FUNCTION(
    void,
    buildFxAndJFxAndSolveRepeatedly,
    (const int iterations),
    "using current data, builds JFx (and Fx) and solves the least squares problem"
    "then does a gradient descent step"
    "reapeats this whole process as often as desired"
    )
{ // TODO we might want to do this externally
    dprintf("buildFxAndJFxAndSolveRepeatedly %d times\n", iterations);
    assert(iterations > 0);
    
    DO(i, iterations) {
        bool buildFx = i == 0; // Fx is always up-to date after first iteration

        buildFxAndJFxAndSolve(buildFx);
        const float delta = addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore();
        if (delta > -0.001) {
            dprintf("delta was only %f, stopping optimization\n", delta);
            return;
        }
    }
}


CPU_FUNCTION(
    void,
    receiveOptimizationDataBuildFxAndJFxAndSolveRepeatedly,
    (
    _In_reads_(xLength) const real* const xI, const size_t xLength,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndicesI, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndicesI, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndicesI, const size_t yIndicesLength,
    const int iterations
    ),
    "Receives x, sparseDerivativeZtoYIndices, xIndices and yIndices"
    "Appropriately sized vectors for receiving these data items is allocated in __managed__ memory, hence this is a CPU only function"
    "currently also builds F(x), JF(x), but that could also be done on the GPU later"
    "it also calls solve, because J is built in local memory so it would be lost later"
    ) {
    receiveOptimizationData(xI, xLength, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength,
        xIndicesI, xIndicesLength, yIndicesI, yIndicesLength);

    buildFxAndJFxAndSolveRepeatedly(iterations);
}

// Prototyping functions

FUNCTION(void,
    receiveAndPrintOptimizationData,
    (
    const int lengthz,
    const int lengthfz,

    _In_reads_(xLength) const real* const x, const size_t xLength,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndices, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndices, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndices, const size_t yIndicesLength
    ),
    "Receives x, sparseDerivativeZtoYIndices, xIndices and yIndices, checks and prints them,"
    "emulating arbitrary lengthz, lengthfz"
    "Note: lengthz, lengthfz are fixed at compile-time for other functions"
    "This is a prototyping function that does not allocate or copy anything"
    "use for testing"
    ) {

    const int lengthP = xIndicesLength / lengthz;
    const int lengthY = yIndicesLength;
    const int lengthFx = lengthfz * lengthP;
    const int maxNNZ = (lengthfz*lengthz) * lengthP; // could go down from lengthz to maximum k in sparseDerivativeZtoYIndices
     // or just the actual sum of all such k

    dprintf("lengthz: %d\n", lengthz);
    dprintf("lengthfz: %d\n", lengthfz);
    dprintf("lengthP: %d\n", lengthP);
    dprintf("lengthY: %d\n", lengthY);
    dprintf("lengthFx: %d\n", lengthFx);
    dprintf("maxNNZ: %d\n", maxNNZ);

    assert(lengthz > 0);
    assert(lengthfz > 0);
    assert(lengthY > 0);

    dprintf("x:\n");
    printv(x, xLength);

    dprintf("sparseDerivativeZtoYIndices:\n");
    const int* p = sparseDerivativeZtoYIndices;
    DO(i, lengthP) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        int k = *p++; 
        assert(k <= lengthz);
        dprintf("---\n");
        printd(p, k); p += k;
        dprintf("-->\n");
        printd(p, k); p += k;
        dprintf("---\n");
    }

    dprintf("xIndices:\n");
    p = xIndices;
    DO(i, lengthP) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        printd(p, lengthz);
        p += lengthz;
    }
    assertEachInRange(xIndices, xIndicesLength, 0, xLength - 1);

    dprintf("yIndices:\n");
    printd(yIndices, yIndicesLength);
    assertEachInRange(yIndices, yIndicesLength, 0, xLength - 1);
}



FUNCTION(
    void,
    makeAndPrintSparseMatrix,
    (
    const size_t m,
    const size_t n,
    _In_reads_(xlen) real* x,
    size_t xlen,
    _In_reads_(ijlen) int* ij,
    const size_t ijlen
    ),
    "Creates a sparse matrix from a list of values and a list of pairs of (i, j) indices specifying where to put the corresponding values. "
    "Note: This is a prototyping function without any further purpose"
    ) {
    assert(2 * xlen == ijlen);
    assert(xlen <= m*n); // don't allow repeated entries

    SOMEMEM();
    cs* const A = cs_spalloc(m, n, xlen, 1, SOMEMEMP);

    while (xlen--) {
        int i = *ij++;
        int j = *ij++;
        cs_entry(A, i, j, *x++);
    }

    cs_print(A);

    printf("compress and print again:\n");
    const cs* const B = cs_triplet(A, SOMEMEMP);
    cs_print(B);
    printf("done--\n");


    FREESOMEMEM();
}








// Misc


FUNCTION(void, testMain, (), "collection of some tests") {
    dprintf("-- sparse optimize solver --\n");
    dprintf("Compiled with:\n");
    dprintf("lengthz: %d\n", lengthz);
    dprintf("lengthfz: %d\n", lengthfz);

    dprintf("POST begins:\n");
    real x[] = {1, 2};
    printv(x, 2);
    real y[] = {1};
    printv(y, 1);

    int to[] = {1};
    axpyWithReindexing(x, 2, 1., y, to, 1); // expect 1.000000 3.000000
    printv(x, 2); 

    real z[] = {0, 0};
    int from[] = {1, 0};
    extract(z, x, 2, from, 2); // expect 3.000000 1.000000
    printv(z, 2);

    // expect i: 0-9
    FOR(int,i,0,10,1)
        dprintf("i: %d\n", i);
}



GLOBAL(int, xx, 0, "xx");
GLOBAL(real, a, 0, "a");
GLOBAL(real, b, 0, "b");
GLOBAL(real, yy, 0, "yy");

FUNCTION(void, xx43, (), "sets xx to 43") {
    xx = 43;
}

FUNCTION(int, get42, (), "returns the answer to the universe life and everything") {
    return 42;
}

FUNCTION(int, add, (int x, int y), "x + y") {
    return x + y;
}

FUNCTION(real, addf, (real x, real y), "x + y with floats") {
    return x + y;
}


FUNCTION(int, multiout, (_Inout_updates_(l) int* x, int l), "return more than one thing") {
    *x = 0;
    return 1;
}
#ifndef WL_WSTP_MAIN 
__global__ void mainc() {

    int cij[] = {0, 0};
    int xlen = 1;
    real xc[] = {0.1};
    real* x = xc;
    int m = 1, n = 1;
    int* ij = cij;

    SOMEMEM();
    cs* A = cs_spalloc(m, n, xlen, 1, SOMEMEMP);

    while (xlen--) {
        int i = *ij++;
        int j = *ij++;
        cs_entry(A, i, j, *x++);
    }

    cs_print(A);

    printf("compress and print again:\n");
    A = cs_triplet(A, SOMEMEMP);
    cs_print(A);


    FREESOMEMEM();
    return;
}

int main() {
    //mainc<<<1,1>>>();
    //hipDeviceSynchronize();
    auto e = 74; printf("hipGetLastError %d %s %s\n", e, hipGetErrorName((hipError_t)e), hipGetErrorString((hipError_t)e));
    return;
}
#endif
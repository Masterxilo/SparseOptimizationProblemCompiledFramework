#include "hip/hip_runtime.h"
/*
This program solves least-squares problems with energies of the form

    \sum_{P \in Q} \sum_{p \in P} ||f(select_p(x))||_2^2

Q gives a partitioning of the domain. In the simplest case, there is only one partition.

The solution to this may or may not be close to the solution to

    \sum_{p \in \Cup Q} ||f(select_p(x))||_2^2

*/

/*
Compile with nvcc, after building the generated files with frameworkBuild.nb
(only needs to rerun when the external interface changes)

To make this CUDA debuggable, be sure to adjust then environment:

NSIGHT_CUDA_DEBUGGER=1

SetEnvironment["NSIGHT_CUDA_DEBUGGER" -> "1"]

Except for paul.h, standard/windows/cuda headers and generated files, this file is self-contained
*/

#define _USE_MATH_DEFINES
#include <math.h>
#include <float.h>
#include <stdio.h>
#include <memory.h> // just memset, but not malloc/free

#define _CRT_SECURE_NO_WARNINGS
#define NOMINMAX
#define WINDOWS_LEAN_AND_MEAN
#include <windows.h>

#include <sal.h> // c.f. sal.txt

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#pragma comment(lib,"cudart")

/*

Paul's CUDA/WSTP-Mathematica/C agnostic framework begins

FUNCTION declared functions can be called from Mathematica and CUDA whenever possible.

CPU_FUNCTION and CUDA_FUNCTION denote functions that use features only available on either platform (e.g. dues to CUDA API limitations or extensions)

Only primitive types can be passed back and forth automatically as of now.

*/


#define WL_WSTP_MAIN        // undefine to use main below to debug without mathematica 
#define WL_ALLOC_CONSOLE
#define WL_WSTP_PRE_MAIN
#include <paulwl.h> 
#include <paul.h>


// Most global data, can be queries via State
#define GLOBAL(type, name, defaultValue, usage) __managed__ type name = defaultValue
#define GLOBALDYNAMICARRAY(elementtype, name, sizevar, usage) __managed__ elementtype* name = 0; __managed__ int sizevar = 0; // lengths are int because that's what WSTP expects -- it doesn't seem to support larger arrays (at least not sent at once) -- should be size_t

#define GLOBALDYNAMICARRAY_SHAREDLENGTH(elementtype, name, sizevar, usage) __managed__ elementtype* name = 0; // TODO would be nice if it was detected that the variable already exists

// Read-only global data
#ifdef __CUDA_ARCH__
#define CONSTANTD const __constant__ 
#else
#define CONSTANTD const 
#endif

#ifdef __CUDA_ARCH__
#define CONSTANT(type, name, value, usage) const __constant__ type name = value
#else
#define CONSTANT(type, name, value, usage) const type name = value
#endif

// Most functions, will have a CUDA-kernel version __global__ void name args, one which outputs to an array from each thread and one that doesn't
#define FUNCTION(ret, name, args, usage) /** usage */ __host__ __device__ ret name args
#define MEMBERFUNCTION(ret, name, args, usage) /** usage */ __host__ __device__ ret name args

// FOR with const iterating variable
// for (int i = 0; i < 10; i++) i+=10; is legal but bad
// FOR(int,i,0,10) i+=10; is illegal

// create a block executed once with decl in place, prefix version of {decl; ...}
#define BLOCK_DECLARE(decl) if (bool a__ = true) for (decl;a__;a__=false)

#define FOR(type, var, start, maxExclusive, inc) for (type var##_ = start; var##_ < maxExclusive; var##_ += inc) BLOCK_DECLARE(const type var = var##_)

#define FOR1(type, var, start, maxExclusive) FOR(type, var, start, maxExclusive, 1)
#define FOR01(type, var, maxExclusive) FOR(type, var, 0, maxExclusive, 1)
#define FOR01S(var, maxExclusive) FOR(size_t, var, 0, maxExclusive, 1)

// Like Mathematica's Do, but starts at 0 and goes to maxExclusive-1
#define DO(var, maxExclusive) FOR01S(var, maxExclusive) MAKE_CONST(var)

#define MAKE_CONST(var) BLOCK_DECLARE(auto var##_ = var) BLOCK_DECLARE(const auto var = var##_)

// Special:

// Denotes a function that uses GPU features (threadIdx, __shared__)
//#define CUDA_FUNCTION(ret, name, args) __device__ ret name args // threadIdx is defined as 1 for CPU functions, __shared__ is ignored

// can call CUDAKERNEL_LAUNCH
#define CPU_FUNCTION(ret, name, args, usage) ret name args





// can theoretically work with any floating point type
// memory alignment issues in CUDA have to be readdressed should this be changed to double
// if this is 8 bytes instead of 4, many nextEven calls could be saved
typedef float real; 






// todo put elsewhere, use CONSTANT() to be able to read them from outside -- this is currently not possible because interplay with the // preprocessor is not implemented in the WSTP wrapper code
/**/
CONSTANTD int lengthz =
#include "lengthz.cpp" /* generated for each problem */
;

CONSTANTD int lengthfz =
#include "lengthfz.cpp" /* generated for each problem */
;


#include "$WSTPWrappingCDefinesCUDA.h" /* generated for interface, used for memory management */
#ifdef WL_WSTP_MAIN
#include "wstpExternC.cu"              /* generated for interface */
#endif

#include "$CFormDefines.cpp"  /* generated for problem, rarely changes */  // Required for including *working* definitions of f and df -- this defines what times(x,y) etc. mean
#define x(i) input[i] /* definitions of f/df use x(i) to refer to input[], c.f. RIFunctionCForm* */

// TODO support these (_In_reads_(lengthz)) kinds of constant-sized vectors in the WSTP wrapper code
// TODO the wstp wrapper code should not be able to produce a caller for this because it does not know the length of input
// should not accept just anything
FUNCTION(void, f, (
    _In_reads_(lengthz) const real* const input,
    _Out_writes_(lengthfz) real* const out
    ), "the local energy vector computing function") {
#include "f.cpp" /* generated for each problem, depends on $CFormDefines*/
}
FUNCTION(void, df, (int const i,
    _In_reads_(lengthz) real const * const input,
    _Out_writes_(lengthfz) real * const out
    ), "the derivatives along the i-th variable of the local energy vector computing function") {
#include "df.cpp" /* generated for each problem */
}
#undef x

/*
CSPARSE
A Concise Sparse Matrix Package in C

http://people.sc.fsu.edu/~jburkardt/c_src/csparse/csparse.html

CSparse Version 1.2.0 Copyright (c) Timothy A. Davis, 2006

reduced to only the things needed for sparse conjugate gradient method

by Paul Frischknecht, August 2016

and for running on CUDA, with a user-supplied memory-pool

modified & used without permission
*/


/* --- primary CSparse routines and data structures ------------------------- */
struct cs    /* matrix in compressed-column or triplet form . must be aligned on 8 bytes */
{
    int nzmax;	/* maximum number of entries allocated for triplet. Actual number of entries for compressed col. > 0 */
    int m;	    /* number of rows > 0 */

    int n;	    /* number of columns  > 0 */
    int nz;	    /* # of entries in triplet matrix, NZ_COMPRESSED_COLUMN_INDICATOR for compressed-col, >= 0 otherwise */

    // Note: this order preserves 8-byte pointer (64 bit) alignment, DO NOT CHANGE
    // all pointers are always valid
    int *p;	    /* column pointers (size n+1) or col indices (size nzmax) */

    int *i;	    /* row indices, size nzmax */

    real *x;	/* numerical values, size nzmax*/

};

FUNCTION(bool,cs_is_triplet,(const cs *A), "whether A is a triplet matrix") {
    assert(A);
    return A->nz >= 0;
}

const int NZ_COMPRESSED_COLUMN_INDICATOR = -1;

FUNCTION(bool,cs_is_compressed_col,(const cs *A),"whether A is a crompressed-column form matrix") {
    assert(A);
    assert(A->m >= 1 && A->n >= 1);
    return A->nz == NZ_COMPRESSED_COLUMN_INDICATOR;
}



// hacky arbitrary memory-management by passing
// reduces memory_size and increases memoryPool on use
#define MEMPOOL char*& memoryPool, int& memory_size // the function taking these modifies memoryPool to point to the remaining free memory
#define MEMPOOLPARAM memoryPool, memory_size 


FUNCTION(char* ,cs_malloc_,(char*& memoryPool, int& memory_size, size_t sz), 
"allocate new stuff. can only allocate multiples of 8 bytes to preserve alignment of pointers in cs. Use nextEven to round up when allocating 4 byte stuff (e.g. int)"){
    assert(memory_size >= sz);
    assert(aligned(memoryPool, 8));
    assert(divisible(sz, 8));
    auto out = memoryPool;
    memoryPool += sz;
    memory_size -= sz;
    return out;
}

#define cs_malloc(varname, sz) {(varname) = (decltype(varname))cs_malloc_(MEMPOOLPARAM, (sz));}

FUNCTION(void ,cs_free_,(char*& memoryPool, int& memory_size, size_t sz) ,"free the last allocated thing of given size"){
    assert(divisible(sz, 8));
    assert(aligned(memoryPool, 8));
    memoryPool -= sz;
    memory_size += sz;
}

#define cs_free(sz) {cs_free_(MEMPOOLPARAM, (sz));}


FUNCTION(int,cs_spalloc_size,(size_t m, size_t n, size_t nzmax, bool triplet),
"amount of bytes a sparse matrix with the given characteristics will occupy"){
    return sizeof(cs) + nextEven(triplet ? nzmax : n + 1) * sizeof(int) + nextEven(nzmax) *  (sizeof(int) + sizeof(real));
}


FUNCTION(cs *,cs_spalloc,(size_t m, size_t n, size_t nzmax, bool triplet, MEMPOOL),
    "allocates a sparse matrix using memory starting at memoryPool,"
    "uses exactly"
    "sizeof(cs) + cs_spalloc_size(m, n, nzmax, triplet) BYTES"
    "of the pool"
    )
{
    char* initial_memoryPool = memoryPool;
    assert(nzmax > 0);

    cs* A; cs_malloc(A, sizeof(cs));    /* allocate the cs struct */

    A->m = m;				    /* define dimensions and nzmax */
    A->n = n;
    A->nzmax = nzmax;
    A->nz = triplet ? 0 : NZ_COMPRESSED_COLUMN_INDICATOR;		    /* allocate triplet or comp.col */

    // Allocate too much to preserve alignment
    cs_malloc(A->p, nextEven(triplet ? nzmax : n + 1) * sizeof(int));
    cs_malloc(A->i, nextEven(nzmax) * sizeof(int));
    cs_malloc(A->x, nextEven(nzmax) * sizeof(real));

    assert(memoryPool == initial_memoryPool + cs_spalloc_size(m, n, nzmax, triplet));
    return A;
}


FUNCTION(int, cs_cumsum, (_Inout_updates_all_(n + 1) int *p, _Inout_updates_all_(n) int *c, const int n),
    "p [0..n] = cumulative sum of c [0..n-1], and then copy p [0..n-1] into c "
    )
{
    assert(n >= 0 && p && c); /* check inputs */
    int i, nz = 0;
    for (i = 0; i < n; i++)
    {
        p[i] = nz;
        nz += c[i];
        c[i] = p[i];
    }
    p[n] = nz;
    return (nz);		    /* return sum (c [0..n-1]) */
}

FUNCTION(int*,allocZeroedIntegers,(const int n, MEMPOOL),"Allocate n integers set to 0. Implements calloc(n, sizeof(int)). n must be even") {
    assert(divisible(n, 2));
    int* w;
    cs_malloc(w, n * sizeof(int));
    memset(w, 0, n*sizeof(int)); // w = (int*)cs_calloc(n, sizeof(int)); /* get workspace */
    return w;
}

// alloc/free a list of integers w, initialized to 0
#define allocTemporaryW(count) int wsz = nextEven((count)); int* w = allocZeroedIntegers(wsz, MEMPOOLPARAM); 
#define freeTemporaryW() cs_free(wsz * sizeof(int)); 


FUNCTION(cs *,cs_transpose,(const cs * const A, MEMPOOL),
    "C = A'"
    ""
    "memoryPool must be big enough to contain the following:"
    "cs_spalloc_size(n, m, Ap[n], 0) --location of output"
    "nextEven(m)*sizeof(int) --temporary")
{
    assert(A && cs_is_compressed_col(A));

    const int m = A->m;
    const int n = A->n;
    int const * const Ai = A->i;
    int const * const Ap = A->p;
    real const * const Ax = A->x;

    cs *C; C = cs_spalloc(n, m, Ap[n], 0, MEMPOOLPARAM); /* allocate result */

    allocTemporaryW(m); /* get workspace */

    int* const Cp = C->p; int* const Ci = C->i; real* const Cx = C->x;
    assert(Cp && Ci && Cx);

    for (int p = 0; p < Ap[n]; p++) w[Ai[p]]++;	   /* row counts */
    cs_cumsum(Cp, w, m);				   /* row pointers */
    for (int j = 0; j < n; j++)
    {
        for (int p = Ap[j]; p < Ap[j + 1]; p++)
        {
            int q;
            Ci[q = w[Ai[p]]++] = j;	/* place A(i,j) as entry C(j,i) */
            Cx[q] = Ax[p];
        }
    }

    freeTemporaryW();

    return C;	/* success; free w and return C */
}

FUNCTION(cs *,cs_triplet,(const cs * const T, MEMPOOL),
    "C = compressed-column form of a triplet matrix T"
    ""
    "memoryPool must be big enough to contain the following"
    "cs_spalloc_size(m, n, nz, 0) --location of output"
    "nextEven(n)* sizeof(int) --temporary")
{
    assert(T && cs_is_triplet(T));/* check inputs */

    const int m = T->m;
    const int n = T->n;
    int const * const Ti = T->i;
    int const * const Tj = T->p;
    real const * const Tx = T->x;
    const int nz = T->nz;

    assert(m > 0 && n > 0);
    cs *C; C = cs_spalloc(m, n, nz, 0, memoryPool, memory_size);		/* allocate result */

    allocTemporaryW(n); /* get workspace */

    int* const Cp = C->p; int* const Ci = C->i; real* const Cx = C->x;
    assert(Cp && Ci && Cx);

    for (int k = 0; k < nz; k++) w[Tj[k]]++;		/* column counts */
    cs_cumsum(Cp, w, n);				/* column pointers */
    for (int k = 0; k < nz; k++)
    {
        int p;
        Ci[p = w[Tj[k]]++] = Ti[k];    /* A(i,j) is the pth entry in C */
        Cx[p] = Tx[k];
    }

    freeTemporaryW();

    return C;	    /* success; free w and return C */
}

FUNCTION(int,cs_entry,(cs * const T, const int i, const int j, const real x),
    "add an entry to a triplet matrix; return 1 if ok, assertion failure otherwise ")
{
    assert(cs_is_triplet(T));
    assert(i >= 0 && i < T->m && j >= 0 && j <= T->n); // cannot enlarge matrix
    assert(T->nz < T->nzmax); // cannot enlarge matrix
    assert(T->x);
    assertFinite(x);

    T->x[T->nz] = x;
    T->i[T->nz] = i;
    T->p[T->nz++] = j;
    return (1);
}


FUNCTION(int,cs_print,(const cs * const A, int brief = 0),"print a sparse matrix")
{
    assert(A);
    int p, j, m, n, nzmax, nz, *Ap, *Ai;
    real *Ax;

    m = A->m; n = A->n; Ap = A->p; Ai = A->i; Ax = A->x;
    nzmax = A->nzmax; nz = A->nz;

    printf("CSparse %s\n",
#ifdef __CUDA_ARCH__
        "on CUDA"
#else
        "on CPU"
#endif
        );
    assert(m > 0 && n > 0);
    if (nz < 0)
    {
        printf("%d-by-%d, nzmax: %d nnz: %d\n", m, n, nzmax,
            Ap[n]);
        for (j = 0; j < n; j++)
        {
            printf("    col %d : locations %d to %d\n", j, Ap[j], Ap[j + 1] - 1);
            for (p = Ap[j]; p < Ap[j + 1]; p++)
            {
                assert(Ai[p] >= 0 && Ai[p] < m);
                printf("      %d : %g\n", Ai[p], Ax ? Ax[p] : 1);
                if (brief && p > 20) { printf("  ...\n"); return (1); }
            }
        }
    }
    else
    {
        printf("triplet: %d-by-%d, nzmax: %d nnz: %d\n", m, n, nzmax, nz);
        assert(nz <= nzmax);
        for (p = 0; p < nz; p++)
        {
            printf("    %d %d : %g\n", Ai[p], Ap[p], Ax ? Ax[p] : 1);
            assert(Ai[p] >= 0 && Ap[p] >= 0);
            if (brief && p > 20) { printf("  ...\n"); return (1); }
        }
    }
    return (1);
}


FUNCTION(int,cs_mv,(real * y, real alpha, const cs *  A, const real * x, real beta),
   "y = alpha A x + beta y"
   "the memory for y and x cannot overlap"
   "TODO implement a version that can transpose A implicitly")
{
    assert(A && x && y);	    /* check inputs */
    assertFinite(beta);
    assertFinite(alpha);

    int p, j, n, *Ap, *Ai;
    real *Ax;
    n = A->n; Ap = A->p; Ai = A->i; Ax = A->x;

    // the height of A is the height of y. Premultiply y with beta, then proceed as before, including the alpha factor when needed 
    // TODO (can we do better?)
    // Common special cases
    if (beta == 0)
        memset(y, 0, sizeof(real) * A->m);
    else
        for (int i = 0; i < A->m; i++) y[i] *= beta;

    if (alpha == 1)
        for (j = 0; j < n; j++) for (p = Ap[j]; p < Ap[j + 1]; p++) y[Ai[p]] += Ax[p] * x[j];
    else if (alpha != 0) // TODO instead of deciding this at runtime, let the developer choose the right function xD
        for (j = 0; j < n; j++) for (p = Ap[j]; p < Ap[j + 1]; p++) y[Ai[p]] += alpha * Ax[p] * x[j];
    // if alpha = 0, we are done

    return (1);
}
// ---


// logging/debugging

GLOBAL(
    int,
    dprintEnabled,
    true,
    "if true, dprintf writes to stdout, otherwise dprintf does nothing"
    "It would be more efficient to compile with dprintf defined to nothing of course"
    "Default: true"
    );

#ifdef __CUDA_ARCH__
#define dprintf(formatstr, ...) {if (dprintEnabled) printf("CUDA " formatstr, __VA_ARGS__);}
#else
#define dprintf(formatstr, ...) {if (dprintEnabled) printf(formatstr, __VA_ARGS__);}
#endif

FUNCTION(void,
    print,
    (_In_z_ const char* const x),
    "prints a string to stdout"){
    printf("print: %s\n", x);
}


FUNCTION(
    void,
    printd,
    (_In_reads_(n) const int* v, size_t n),
    "dprints a vector of integers, space separated and newline terminated"
    ) {
    while (n--) dprintf("%d ", *v++); dprintf("\n");
}

FUNCTION(
    void,
    printJ,
    (cs* J),
    "prints a sparse matrix"
    ){
    if (dprintEnabled) cs_print(J);
}


// for conjgrad/sparse leastsquares:

// infrastructure like CusparseSolver

/*
Implementation note:
size_t n is not const because the implementation modifies it

changes to n are not visible outside anyways:
-> the last const spec is always an implementation detail, not a promise to the caller, but can indicate conceptual thinking
*/
FUNCTION(
    void,
    printv,
    (_In_reads_(n) const real* v, size_t n),
    "dprints a vector of doubles, space separated and newline terminated"
    ) {
    while (n--) dprintf("%f ", *v++); dprintf("\n");
}

struct vector {
    real* x;
    int n;

    MEMBERFUNCTION(void,print,(),"print this vector") {
        printv(x, n);
    }
};

FUNCTION(void, assertFinite, (_In_reads_(n) const real* const x, const int n), "assert that each element in v is finite") {
    for (int i = 0; i < n; i++)
        assertFinite(x[i]);
}

FUNCTION(void,assertFinite,(const vector& const v),"assert that each element in v is finite") {
    assertFinite(v.x, v.n);
}

FUNCTION(vector,vector_wrapper,(real* x, int n),"create a vector object pointing to existing memory for convenient accessing") {
    vector v;
    v.n = n;
    v.x = x;
    assertFinite(v);
    return v;
}

FUNCTION(vector,vector_allocate,(int n, MEMPOOL),"Create a new vector. uninitialized: must be written before it is read!") {
    vector v;
    v.n = n;
    cs_malloc(v.x, sizeof(real) * nextEven(v.n));
    return v;
}

FUNCTION(vector,vector_copy,(const vector& other, MEMPOOL),"create a copy of other") {
    vector v;
    v.n = other.n;
    cs_malloc(v.x, sizeof(real) * nextEven(v.n));
    memcpy(v.x, other.x, sizeof(real) * v.n);
    assertFinite(v);
    return v;
}

struct matrix {
    const cs* const mat; // in compressed column form (transpose does not work with triplets)

    __declspec(property(get = getRows)) int rows;
    MEMBERFUNCTION(int,getRows,(),"m") const {
        return mat->m;
    }
    __declspec(property(get = getCols)) int cols;
    MEMBERFUNCTION(int, getCols, (), "n") const {
        return mat->n;
    }


    MEMBERFUNCTION(,matrix,(const cs* const mat),"construct a matrix wrapper") : mat(mat) {
        assert(!cs_is_triplet(mat));
        assert(mat->m && mat->n);
        assertFinite(mat->x, mat->nz);
    }

    MEMBERFUNCTION(void,print,() ,"print this matrix"){
        cs_print(mat, 0);
    }
};


FUNCTION(real,dot,(const vector& x, const vector& y), "result = <x, y>, aka x.y or x^T y (the dot-product of x and y)"){
    assert(y.n == x.n);
    real r = 0;
    for (int i = 0; i < x.n; i++) r += x.x[i] * y.x[i];
    return r;
}

FUNCTION(void,axpy,(vector& y, const real alpha, const vector& x),"y = alpha * x + y") {
    assert(y.n == x.n);
    for (int i = 0; i < x.n; i++) y.x[i] += alpha * x.x[i];
}

FUNCTION(void,axpy,(vector& y, const vector& x),"y = x + y") {
    axpy(y, 1, x);
}

FUNCTION(void,scal,(vector& x, const real alpha),"x *= alpha"){
    for (int i = 0; i < x.n; i++) x.x[i] *= alpha;
}

FUNCTION(void,mv,(vector& y, const real alpha, const matrix& A, const vector& x, const real beta),
"y = alpha A x + beta y"){
    assert(A.mat->m && A.mat->n);
    assert(y.n == A.mat->m);
    assert(x.n == A.mat->n);
    cs_mv(y.x, alpha, A.mat, x.x, beta);
}

FUNCTION(void,mv,(vector& y, const matrix& A, const vector& x), "y = A x"){
    mv(y, 1, A, x, 0);
}

FUNCTION(matrix,transpose,(const matrix& A, MEMPOOL),"A^T") {
    return matrix(cs_transpose(A.mat, MEMPOOLPARAM));
}

#define memoryPush() const auto old_memoryPool = memoryPool; const auto old_memory_size = memory_size; //savepoint: anything allocated after this can be freed again
#define memoryPop() {memoryPool = old_memoryPool; memory_size = old_memory_size;} // free anything allocated since memory push

// core algorithm, adapted from CusparseSolver, originally copied from wikipedia
/* required operations:
- new vector of given size
- copy/assign vector
- mv_T, mv (matrix (transpose) times vector) -- because I did not come up with a transposing-multiply operation, I just compute AT once instead of using mv_T
- scal (scaling)
- axpy // y = alpha * x + y
*/
//function [x] = conjgrad_normal(A,b,x)
/*The conjugate gradient method can be applied to an arbitrary n-by-m matrix by applying it to normal equations ATA and right-hand side vector ATb, since ATA is a symmetric positive-semidefinite matrix for any A. The result is conjugate gradient on the normal equations (CGNR).
ATAx = ATb
As an iterative method, it is not necessary to form ATA explicitly in memory but only to perform the matrix-vector and transpose matrix-vector multiplications.

x is an n-vector in this case still

x is used as the initial guess -- it may be 0 but must in any case contain valid numbers
*/
FUNCTION(void,conjgrad_normal,(
    const matrix& A,
    const vector& b,
    vector& x,
    MEMPOOL),
    "x = A\b"
    )
    {
    memoryPush(); //savepoint: anything allocated after this can be freed again

    int m = A.rows, n = A.cols;

    matrix AT = transpose(A, MEMPOOLPARAM); // TODO implement an mv that does transposing in-place

    vector t = vector_allocate(m, MEMPOOLPARAM);

    vector r = vector_allocate(n, MEMPOOLPARAM); mv(r, AT, b); mv(t, A, x); mv(r, -1, AT, t, 1);//r=A^T*b; t = A*x; r = -A^T*t + r;//r=A^T*b-A^T*A*x;

    vector p = vector_copy(r, MEMPOOLPARAM);//p=r;

    real rsold = dot(r, r);//rsold=r'*r;
    if (sqrt(rsold) < 1e-5) goto end; // low residual: solution found

    vector Ap = vector_allocate(A.cols, MEMPOOLPARAM);

    for (int i = 1; i <= b.n; i++) {
        mv(t, A, p); mv(Ap, AT, t);//t = A*p;Ap=A^T*t;//Ap=A^T*A*p;

        if (abs(dot(p, Ap)) < 1e-9) { printf("conjgrad_normal emergency exit\n"); break; }// avoid almost division by 0
        real alpha = rsold / (dot(p, Ap));//alpha=rsold/(p'*Ap);

        axpy(x, alpha, p);//x = alpha p + x;//x=x+alpha*p;
        axpy(r, -alpha, Ap);//r = -alpha*Ap + r;//r=r-alpha*Ap;
        real rsnew = dot(r, r);//rsnew=r'*r;
        if (sqrt(rsnew) < 1e-5) break; // error tolerance, might also limit amount of iterations or check change in rsnew to rsold...
        real beta = (rsnew / rsold);
        scal(p, beta); axpy(p, r);//p*=(rsnew/rsold); p = r + p;//p=r+(rsnew/rsold)*p;
        rsold = rsnew;//rsold=rsnew;
    }

end:
    memoryPop(); // free anything allocated since memory push
}

// solving least-squares problems
FUNCTION(int, cs_cg, (const cs * const A, _In_reads_(A->m) const real * const b, _Inout_updates_all_(A->n) real *x, MEMPOOL),
"x=A\b"
"current value of x is used as initial guess"
"Uses memory pool to allocate transposed copy of A and four vectors with size m or n")
{
    assert(A && b && x && memoryPool && memory_size > 0);

    auto xv = vector_wrapper(x, A->n);
    conjgrad_normal(matrix(A), vector_wrapper((real*)b, A->m), xv, MEMPOOLPARAM);

    return 1;
}

/*

CSPARSE library end

*/



#define copy mallocmemcpy

FUNCTION(int, lengthzGet, (), "lengthz"){ return lengthz; }
FUNCTION(int, lengthfzGet, (), "lengthfz") { return lengthfz; }


// pure functions, utilities

FUNCTION(
    void,
    assertEachInRange,
    (
    _In_reads_(len) const int* v, 
    size_t len, 
    const int min,
    const int max
    ),
    "computes the same as BoolEval[min <= v <= max]"
    ) {
    assert(v);
    while (len--) { // Note: len reduced once more, gets gigantic if len was already 0
        assert(min <= *v && *v <= max);
        v++;
    }
}

FUNCTION(
    void,

    axpyWithReindexing,

    (
    _Inout_updates_(targetLength) real* const targetBase,
    const size_t targetLength,
    real const a,

    _In_reads_(targetIndicesAndAddedValuesLength) const real* const addedValues,
    _In_reads_(targetIndicesAndAddedValuesLength) const int* const targetIndices,
    const size_t targetIndicesAndAddedValuesLength
    ),
    "targetBase[[targetIndices]] += a * addedValues. Repeated indices are not supported, so addedValues cannot be longer than the target."
    "Note that not necessarily all of target is updated (_Inout_updates_, not _Inout_updates_all_)"

    ) {
    assert(targetLength); // targetLength - 1 overflows otherwise
    assertFinite(a);
    assert(targetIndicesAndAddedValuesLength <= targetLength);
    dprintf("axpyWithReindexing %f %d %d\n", a, targetLength, targetIndicesAndAddedValuesLength);

    dprintf("target before:\n"); printv(targetBase, targetLength);
    dprintf("targetIndices:\n"); printd(targetIndices, targetIndicesAndAddedValuesLength);
    dprintf("addedValues:\n"); printv(addedValues, targetIndicesAndAddedValuesLength);
    
    assertEachInRange(targetIndices, targetIndicesAndAddedValuesLength, 0, targetLength - 1);

    DO(j, targetIndicesAndAddedValuesLength)
        assertFinite(targetBase[targetIndices[j]] += addedValues[j] * a);

    dprintf("target after:\n"); printv(targetBase, targetLength);
}

FUNCTION(void, extract, (
    _Out_writes_all_ (sourceIndicesAndTargetLength) real* const target,

    _In_reads_(sourceLength) const real* const source,
    const size_t sourceLength,

    _In_reads_(sourceIndicesAndTargetLength) const int* const sourceIndices,
    const size_t sourceIndicesAndTargetLength
    ),
    "target = source[[sourceIndices]]. Note that all of target is initialized (_Out_writes_all_)"
    ) {
    assertEachInRange(sourceIndices, sourceIndicesAndTargetLength, 0, sourceLength - 1);

    DO(i, sourceIndicesAndTargetLength)
        target[i] = source[sourceIndices[i]];
}

// SparseOptimizationProblem library

// --- Memory pool passed to the csparse library ---

// this is ideally some __shared__ memory in CUDA: In CUDA (I think) 
// C-style "stack" memory is first register based but then spills to main memory
// (is shared memory also used for the registers? Just another way to access the register file?)
// this memory does not need to be manually freed

// DEBUG TODO moved memory to global space for debugging -- move to __shared__ again.
// down the stack, no two functions should be calling SOMEMEM at the same time!

//__managed__ char memory[40000/*"Maximum Shared Memory Per Block" -> 49152*/ * 1000]; // TODO could allocate 8 byte sized type, should be aligned then (?)
//__managed__ bool claimedMemory = false; // makes sure that SOMEMEM is only called by one function on the stack

// "A default heap of eight megabytes is allocated if any program uses malloc() without explicitly specifying the heap size." -- want more 

void preWsMain() { // using a constructor to do this seems not to work
    int const mb = 100;
    dprintf("setting cuda malloc heap size to %d mb\n", mb);
    hipDeviceSetLimit(hipLimitMallocHeapSize, mb * 1000 * 1000); // basically the only memory we will use, so have some!
    CUDA_CHECK_ERRORS();
}

#define SOMEMEM() \
    const size_t memory_size = 4000  * 1000;\
    char* const memory = (char*)malloc(memory_size);/*use global memory afterall*/\
    char* mem = (char*)(((unsigned long long)memory+7) & (~ 0x7ull)); /* align on 8 byte boundary */\
    assert(aligned(mem, 8) && after(mem, memory));\
    int memsz = memory_size - 8;/*be safe*/ \
    assert(memsz>0);\
    bool claimedMemory = true;\
    dprintf("allocated %d bytes at %p using malloc\n", memory_size, memory);\
    assert(memory); /*attempting to access a null pointer just gives a kernel launch failure on GPU most of the time - at least when debugger cannot be attached */

#define FREESOMEMEM() {assert(claimedMemory); claimedMemory = false; free(memory); mem = 0;}


#define SOMEMEMP mem, memsz

// --- end of memory pool stuff ---


// one separate SOP (for one P in Q), shares only "x" with the global problem
// has custom y, p and values derived from that
// pointers are to __managed__ memory
// F() is another function for each partition P. It is defined as (f(s_p(x)))_{p in P}
struct SOPPartition {
    real* minusFx; size_t lengthFx; // "-F(x)"
    real* h; size_t lengthY; // "h, the update to y (subset of x, the parameters currently optimized over)"

    /*
    "amount of 'points' at which the function f is evaluated."
    "lengthP * lengthz is the length of xIndices, "
    "and sparseDerivativeZtoYIndices contains lengthP sequences of the form (k [k many z indices] [k many y indices]) "
    */
    size_t lengthP;

    // integer matrix of dimensions lengthz x lengthP, indexing into x to find the values to pass to f
    int* xIndices;

    // Used to construct J, c.f. SOPJF
    int* sparseDerivativeZtoYIndices; // serialized form of this ragged array

    /*
    "the indices into x that indicate where the y are"
    "needed to write out the final update h to the parameters"
    */
    int* yIndices; /* lengthY */
};



GLOBALDYNAMICARRAY(
    SOPPartition, partitionTable, partitions,
    "partitions of the SOPD, allocated on call to setPartitions");

FUNCTION(void, writeJFx, (cs* const J, const size_t i, const size_t j, const real x),
    "set J(i, j) = x"
    ) {
    assert(J);
    assert(cs_is_triplet(J));
    assert(i < J->m && j < J->n);
    assert(J->nz + 1 <= J->nzmax); // matrix should not become overful
    assertFinite(x);

    cs_entry(J, i, j, x);
}

FUNCTION(void, writeFx, (SOPPartition* const sop, const size_t i, const real val), "F(x)_i = val") {
    assert(i < sop->lengthFx);
    assert(sop->minusFx);
    assertFinite(val);

    sop->minusFx[i] = -val;
}

GLOBALDYNAMICARRAY(
    real, x, lengthx,
    "stores the current data vector 'x' which is updated to reduce the energy ||F(x)||^2"
    );



// -----------------------
/*
Given access to :

int lengthP
int lengthY
const int lengthz (compile-time constant)
const int lengthfz (compile-time constant)
f(fz_out, z)
df(i, fz_out, z)
real* x
int* xIndices (a list of indices into x, lengthfz * n many)
int* sparseDerivativeZtoYIndices (a list of n lists of integers of the structure {k   (deriveby - k integers from 0 to argcount(f)-1) (store at - k integers from 0 to y_length-1)

This creates the vector
Fx
and the sparse matrix
JFx

By calling

void writeFx(int i, real val)
void writeJFx(int i, int j, real val)

using only elementary C constructs
*/
// TODO move these functions to SOPPartition instead of passing the pointer all the time
FUNCTION(void, readZ, (
    SOPPartition* const sop,
    _Out_writes_all_(lengthz) real* z,
    const size_t rowz
    ), "z = x[[xIndices[[rowz;;rowz+lengthz-1]]]]"){
    assert(divisible(rowz, lengthz));

    extract(z, x, lengthx, sop->xIndices + rowz, lengthz); // z = x[[xIndices]]
}

FUNCTION(void, readZandSetFxRow, (
    SOPPartition* const sop,
    _Out_writes_all_(lengthz) real* z,
    const size_t rowz,
    const size_t rowfz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]] = f(z) and return the z = x[[xIndices[[rowz;;rowz+lengthz-1]]]] required for that"){
    assert(divisible(rowz, lengthz));
    assert(divisible(rowfz, lengthfz));

    readZ(sop, z, rowz); // z = x[[xIndices]]

    real fz[lengthfz];
    f(z, fz); // fz = f(z)

    DO(i, lengthfz) writeFx(sop, rowfz + i, fz[i]); // Fx[[rowfz;;rowfz+lengthfz-1]] = fz
}

FUNCTION(void, setFxRow, (
    SOPPartition* const sop,
    const size_t rowz,
    const size_t rowfz
    ), "compute and store Fx[[rowfz;;rowfz+lengthfz-1]]"){
    real z[lengthz];
    readZandSetFxRow(sop, z, rowz, rowfz);
}

FUNCTION(void, buildFx, (SOPPartition* const sop), "from the current x, computes just F(x)"){
    size_t rowz = 0;
    size_t rowfz = 0;

    FOR(size_t, i, 0, sop->lengthP, (rowz += lengthz, rowfz += lengthfz, 1)) MAKE_CONST(rowz) MAKE_CONST(rowfz) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        setFxRow(sop, rowz, rowfz);
    }
}

FUNCTION(void, buildFxandJFx, (SOPPartition* const sop, cs* const J, bool buildFx),
    "from the current x, computes F(x) [if buildFx == true] and JF(x)"
    "Note that J is stored into the matrix pointed to"
    "this J must by in triplet form and have allocated enough space to fill in the computed df"
    ) {
    assert(cs_is_triplet(J));
    auto* currentSparseDerivativeZtoYIndices = sop->sparseDerivativeZtoYIndices;
    size_t rowz = 0;
    size_t rowfz = 0;

    FOR(size_t, i, 0, sop->lengthP, (rowz += lengthz, rowfz += lengthfz, 1)) MAKE_CONST(rowz) MAKE_CONST(rowfz) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        real z[lengthz];
        if (buildFx)
            readZandSetFxRow(sop, z, rowz, rowfz);
        else
            readZ(sop, z, rowz);

        // deserialize sparseDerivativeZtoYIndices, c.f. flattenSparseDerivativeZtoYIndices
        // convert back to two lists of integers of the same length (K)
        const int K = *currentSparseDerivativeZtoYIndices++;
        assert(K <= lengthz);
        const int* const zIndices = currentSparseDerivativeZtoYIndices; currentSparseDerivativeZtoYIndices += K;
        const int* const yIndices = currentSparseDerivativeZtoYIndices; currentSparseDerivativeZtoYIndices += K;

        // construct & insert localJ columnwise
        DO(k, K) {
            const int zIndex = zIndices[k];
            const int yIndex = yIndices[k];

            assert(zIndex < lengthz);
            assert(yIndex < sop->lengthY);

            real localJColumn[lengthfz];
            df(zIndex, z, localJColumn);

            // put in the right place (starting at rowfz, column yIndex)
            DO(j, lengthfz) {
                writeJFx(J, rowfz + j, yIndex, localJColumn[j]);
            }
        }
    }
}
// -----------------------

// Core algorithms

FUNCTION(void,
    solve,
    (SOPPartition* const sop, cs const * const J, MEMPOOL),
    "assumes x, -Fx and J have been built"
    "computes the adjustment vector h, which is the least-squares solution to the system"
    "Jh = -Fx"
    ) {
    assert(J && x && sop && sop->minusFx && sop->h);
    assert(cs_is_compressed_col(J));

    dprintf("sparse leastSquares (cg) %d x %d... (this might take a while)\n",
        J->m, J->n);

    assert(sop->lengthY > 0);

    // h must be initialized -- initial guess -- use 0
    memset(sop->h, 0, sizeof(real) * sop->lengthY); // not lengthFx! -- in page writing error -- use struct vector to keep vector always with its length (existing solutions?)

    cs_cg(J, sop->minusFx, sop->h, MEMPOOLPARAM);

    dprintf("h:\n"); printv(sop->h, sop->lengthY);
    assertFinite(sop->h, sop->lengthY);
}

FUNCTION(
    real,
    norm2Fx,
    (SOPPartition const * const sop), "Assuming F(x) is computed, returns ||F(x)||_2^2"
    ) {
    assert(sop->minusFx);
    real x = 0;
    DO(i, sop->lengthFx) x += sop->minusFx[i] * sop->minusFx[i];
    return assertFinite(x);
}

FUNCTION(
    float,
    addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore,
    (SOPPartition * const sop),
    "scales h such that F(x0 + h) < F(x) in the 2-norm and updates x = x0 + h"
    "returns total energy delta achieved which should be negative but might not be when the iteration count is exceeded"
    ) {
    assert(sop);
    assert(sop->yIndices);
    assert(sop->minusFx);
    assert(x);

    // determine old norm
    const real norm2Fatx0 = norm2Fx(sop);
    dprintf("||F[x0]||_2^2 = %f\n", norm2Fatx0);

    // add full h
    real lambda = 1.;
    dprintf("x = "); printv(x, lengthx);
    axpyWithReindexing(x, lengthx, lambda, sop->h, sop->yIndices, sop->lengthY); // xv = x0 + h
    dprintf("x = x0 + h = "); printv(x, lengthx); 

    buildFx(sop);
    real norm2Faty0 = norm2Fx(sop);
    dprintf("||F[x0 + h]||_2^2 = %f\n", norm2Faty0);


    // Reduce step-size if chosen step does not lead to reduction by subtracting lambda * h
    size_t n = 0; // safety net, limit iterations
    while (norm2Faty0 > norm2Fatx0 && n++ < 20) {
        lambda /= 2.;
        axpyWithReindexing(x, lengthx, -lambda, sop->h, sop->yIndices, sop->lengthY); // xv -= lambda * h // note the -!

        dprintf("x = "); printv(x, lengthx); 

        buildFx(sop); // rebuild Fx after this change to x
        norm2Faty0 = norm2Fx(sop); // reevaluate norm
        dprintf("reduced stepsize, lambda =  %f, ||F[y0]||_2^2 = %f\n", lambda, norm2Faty0);
    }
    dprintf("optimization finishes, total energy change: %f\n", norm2Faty0-norm2Fatx0);
    /*assert(norm2Faty0 - norm2Fatx0 <= 0.);*/ // might not be true if early out was used
    return norm2Faty0 - norm2Fatx0;
}

// Interface

FUNCTION(
    void,
    getY,
    (
    int partition,
    _Out_writes_all_(lengthY) real* const outY,
    int lengthY
    ),
    "Extracts from x the elements that are y"
    "lengthY must match the length passed at initialization"
    "TODO make WSTP support having global variables specify the necessary array length, support non size_t lengths"
    "then we wouldn't need to supply the redundant lengthY here"
    )
{
    assert(partition >= 0 && partition < partitions);
    const SOPPartition* const sop = &partitionTable[partition];
    assert(x && sop && sop->yIndices && lengthx && sop->lengthY);
    assert(sop->lengthY <= lengthx);
    assert(lengthY == sop->lengthY);
    extract(outY, ::x, ::lengthx, sop->yIndices, sop->lengthY);
}

// TODO size_t not properly supported because of WSTP receiving only Integer64
CPU_FUNCTION(
    void, setPartitions, (size_t newPartitionsCount), "set the amount of partitions") {

    // free old stuff first
    assert(partitions >= 0);
    DO(i, partitions) {
        SOPPartition* const sop = &partitionTable[i];

        memoryFree(sop->sparseDerivativeZtoYIndices);
        memoryFree(sop->xIndices);
        memoryFree(sop->yIndices);
        memoryFree(sop->minusFx);
        memoryFree(sop->h);
    }
    memoryFree(partitionTable);

    // allocate
    ::partitions = newPartitionsCount;
    partitionTable = tmalloczeroed<SOPPartition>(partitions); // pointers not yet initialized
}

CPU_FUNCTION(
    void,
    receiveSharedOptimizationData,
    (
    _In_reads_(xLength) const real* const xI, const size_t xLength
    ),
    "Receives x"
    ) {
    memoryFree(x);
    ::x = copy(xI, xLength);
    ::lengthx = xLength;
}

// macro for indexing into partitionTable, sop = partitionTable[partition]
#define extractSop(partition) assert(partition >= 0 && partition < partitions); SOPPartition* const sop = &partitionTable[partition];

CPU_FUNCTION(
    void,
    receiveOptimizationData,
    (
    const int partition,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndicesI, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndicesI, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndicesI, const size_t yIndicesLength
    ),
    "Receives sparseDerivativeZtoYIndices, xIndices and yIndices"
    "Appropriately sized vectors for receiving these data items are newly allocated in __managed__ memory, hence this is a CPU only function"
    ) {
    extractSop(partition);

    sop->sparseDerivativeZtoYIndices = copy(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength);
    sop->xIndices = copy(xIndicesI, xIndicesLength);
    sop->yIndices = copy(yIndicesI, yIndicesLength);

    assert(lengthz > 0);
    assert(divisible(xIndicesLength, lengthz));
    assert(lengthfz > 0);

    sop->lengthP = xIndicesLength / lengthz;
    sop->lengthY = yIndicesLength;
    sop->lengthFx = lengthfz * sop->lengthP;

    sop->minusFx = tmalloc<real>(sop->lengthFx);

    sop->h = tmalloc<real>(sop->lengthY);
}

FUNCTION(
    void,
    buildFxAndJFxAndSolve,
    (SOPPartition * const sop, bool buildFx),
    "using current data, builds JFx (and Fx) and solves the least squares problem"
    "optionally does not compute Fx, assuming it is current with the x data (true after every solve)"
    ""
    "Note that we must do the solving right here, because this function handles the memory needed by J"
    "the solution is then accessible in h for further processing (updating x at yIndices)"
    ""
    "sop is passed here, not partition. Use buildFxAndJFxAndSolveRepeatedly as the external interface"
    )
{
    // Build F and JF
    const size_t maxNNZ = (lengthfz*lengthz) * sop->lengthP; // very pessimistic estimate/overestimation: assume every derivative figures for every P -- usually not all of them will be needed
    // ^^ e.g. in vsfs the 3 color channels are all not optimized over, neither doriginal

    // consider using dynamic allocation in SOMEMEM!

    SOMEMEM(); 
    dprintf("allocating sparse matrix for %d entries\n", maxNNZ);
    cs* J = cs_spalloc(sop->lengthFx, sop->lengthY, maxNNZ, 1, SOMEMEMP); // might run out of memory here

    dprintf("buildFxandJFx\n");
    buildFxandJFx(sop, J, buildFx);

    dprintf("used %d of %d allocated spaces in J\n", J->nz, J->nzmax);
    assert(J->nz > 0); // there must be at least one (nonzero) entry in the jacobian, otherwise we have taken the derivative only over variables no ocurring (or no variables at all!)

    J = cs_triplet(J, SOMEMEMP); // "optimizes storage of J, after which it may no longer be modified" 
    // TODO recycle memory

    // State
    dprintf("-F(x):\n");
    printv(sop->minusFx, sop->lengthFx);
    dprintf("JF(x):\n");
    printJ(J);

    // Solve
    dprintf("solve:\n");
    solve(sop, J, SOMEMEMP); // TODO allocates even more memory

    FREESOMEMEM();
}

FUNCTION(
    void,
    buildFxAndJFxAndSolveRepeatedly,
    (const int partition, const int iterations),
    "using current data, builds JFx (and Fx) and solves the least squares problem"
    "then does a gradient descent step"
    "reapeats this whole process as often as desired"
    )
{ 
    extractSop(partition);

    // TODO we might want to do this externally
    dprintf("\n=== buildFxAndJFxAndSolveRepeatedly %d times in partition %d  ===\n", iterations, partition);
    assert(iterations > 0); // TODO iterations should be size_t
    
    DO(i, iterations) {
        bool buildFx = i == 0; // Fx is always up-to date after first iteration

        buildFxAndJFxAndSolve(sop, buildFx);
        const float delta = addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore(sop);
        if (delta > -0.001) {
            dprintf("delta was only %f, stopping optimization\n", delta);
            return;
        }
    }
}

FUNCTION(
    void,
    buildFxAndJFxAndSolveRepeatedlyThreadIdPartition,
    (const int iterations),
    "buildFxAndJFxAndSolveRepeatedly on the partition given by linear_global_threadId."
    "does nothing when linear_global_threadId is >= partitions"
    ""
    "TODO this should be the block id, threads in the same block should cooperate in the same partition"
    )
{
    if (linear_global_threadId() >= partitions) {
        dprintf("\n--- thread id %d has nothing to do  - there are only %d partitions\n", linear_global_threadId(), partitions);
        return;
    }

    dprintf("\n=== Starting work on partition %d in the thread of the same id ===\n", linear_global_threadId());
    buildFxAndJFxAndSolveRepeatedly(linear_global_threadId(), iterations);
}

// Prototyping functions

FUNCTION(void,
    receiveAndPrintOptimizationData,
    (
    const int lengthz,
    const int lengthfz,

    _In_reads_(xLength) const real* const x, const size_t xLength,
    _In_reads_(sparseDerivativeZtoYIndicesLength) const int* const sparseDerivativeZtoYIndices, const size_t sparseDerivativeZtoYIndicesLength,
    _In_reads_(xIndicesLength) const int* const xIndices, const size_t xIndicesLength,
    _In_reads_(yIndicesLength) const int* const yIndices, const size_t yIndicesLength
    ),
    "Receives x, sparseDerivativeZtoYIndices, xIndices and yIndices, checks and prints them,"
    "emulating arbitrary lengthz, lengthfz"
    "Note: lengthz, lengthfz are fixed at compile-time for other functions"
    "This is a prototyping function that does not allocate or copy anything"
    "use for testing"
    ) {

    const int lengthP = xIndicesLength / lengthz;
    const int lengthY = yIndicesLength;
    const int lengthFx = lengthfz * lengthP;
    const int maxNNZ = (lengthfz*lengthz) * lengthP; // could go down from lengthz to maximum k in sparseDerivativeZtoYIndices
     // or just the actual sum of all such k

    dprintf("lengthz: %d\n", lengthz);
    dprintf("lengthfz: %d\n", lengthfz);
    dprintf("lengthP: %d\n", lengthP);
    dprintf("lengthY: %d\n", lengthY);
    dprintf("lengthFx: %d\n", lengthFx);
    dprintf("maxNNZ: %d\n", maxNNZ);

    assert(lengthz > 0);
    assert(lengthfz > 0);
    assert(lengthY > 0);

    dprintf("x:\n");
    printv(x, xLength);

    dprintf("sparseDerivativeZtoYIndices:\n");
    const int* p = sparseDerivativeZtoYIndices;
    DO(i, lengthP) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        int k = *p++; 
        assert(k <= lengthz);
        dprintf("---\n");
        printd(p, k); p += k;
        dprintf("-->\n");
        printd(p, k); p += k;
        dprintf("---\n");
    }
    assert(p == sparseDerivativeZtoYIndices + sparseDerivativeZtoYIndicesLength);

    dprintf("xIndices:\n");
    p = xIndices;
    DO(i, lengthP) {
        DBG_UNREFERENCED_LOCAL_VARIABLE(i);
        printd(p, lengthz);
        p += lengthz;
    }
    assert(p == xIndices + xIndicesLength);
    assertEachInRange(xIndices, xIndicesLength, 0, xLength - 1);

    dprintf("yIndices:\n");
    printd(yIndices, yIndicesLength);
    assertEachInRange(yIndices, yIndicesLength, 0, xLength - 1);
}



FUNCTION(
    void,
    makeAndPrintSparseMatrix,
    (
    const size_t m,
    const size_t n,
    _In_reads_(xlen) real* x,
    size_t xlen,
    _In_reads_(ijlen) int* ij,
    const size_t ijlen
    ),
    "Creates a sparse matrix from a list of values and a list of pairs of (i, j) indices specifying where to put the corresponding values (triplet form)"
    "Note: This is a prototyping function without any further purpose"
    ) {
    assert(2 * xlen == ijlen);
    assert(xlen <= m*n); // don't allow repeated entries

    SOMEMEM();
    cs* const A = cs_spalloc(m, n, xlen, 1, SOMEMEMP);

    while (xlen--) {
        int i = *ij++;
        int j = *ij++;
        cs_entry(A, i, j, *x++);
    }

    cs_print(A);

    printf("compress and print again:\n");
    const cs* const B = cs_triplet(A, SOMEMEMP);
    cs_print(B);
    printf("done--\n");


    FREESOMEMEM();
}








// Misc


FUNCTION(void, testMain, (), "collection of some tests") {
    dprintf("-- sparse optimize solver --\n");
    dprintf("Compiled with:\n");
    dprintf("lengthz: %d\n", lengthz);
    dprintf("lengthfz: %d\n", lengthfz);

    dprintf("POST begins:\n");
    real x[] = {1, 2};
    printv(x, 2);
    real y[] = {1};
    printv(y, 1);

    int to[] = {1};
    axpyWithReindexing(x, 2, 1., y, to, 1); // expect 1.000000 3.000000
    printv(x, 2); 

    real z[] = {0, 0};
    int from[] = {1, 0};
    extract(z, x, 2, from, 2); // expect 3.000000 1.000000
    printv(z, 2);

    // expect i: 0-9
    FOR(int,i,0,10,1)
        dprintf("i: %d\n", i);
}



GLOBAL(int, xx, 0, "xx");
GLOBAL(real, a, 0, "a");
GLOBAL(real, b, 0, "b");
GLOBAL(real, yy, 0, "yy");

FUNCTION(void, xx43, (), "sets xx to 43") {
    xx = 43;
}

FUNCTION(int, get42, (), "returns the answer to the universe life and everything") {
    return 42;
}

FUNCTION(int, add, (int x, int y), "x + y") {
    return x + y;
}

FUNCTION(real, addf, (real x, real y), "x + y with floats") {
    return x + y;
}


FUNCTION(int, multiout, (_Inout_updates_(l) int* x, int l), "returns more than one thing: when called via WSTP this will return an Association with all results") {
    *x = 0;
    return 1;
}
#ifndef WL_WSTP_MAIN 
__global__ void mainc() {

    int cij[] = {0, 0};
    int xlen = 1;
    real xc[] = {0.1};
    real* x = xc;
    int m = 1, n = 1;
    int* ij = cij;

    SOMEMEM();
    cs* A = cs_spalloc(m, n, xlen, 1, SOMEMEMP);

    while (xlen--) {
        int i = *ij++;
        int j = *ij++;
        cs_entry(A, i, j, *x++);
    }

    cs_print(A);

    printf("compress and print again:\n");
    A = cs_triplet(A, SOMEMEMP);
    cs_print(A);


    FREESOMEMEM();
    return;
}

int main() {
    //mainc<<<1,1>>>();
    //hipDeviceSynchronize();
    auto e = 74; printf("hipGetLastError %d %s %s\n", e, hipGetErrorName((hipError_t)e), hipGetErrorString((hipError_t)e));
    return;
}
#endif
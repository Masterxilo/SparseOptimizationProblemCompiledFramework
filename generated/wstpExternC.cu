#include "hip/hip_runtime.h"
void setPartitions(size_t newPartitionsCount);

extern "C" void setPartitions_()
{
WSGet(long long, newPartitionsCount, Integer64);
setPartitions(newPartitionsCount);
WL_RETURN_VOID();
}

void receiveSharedOptimizationData(real const * const xI, size_t const xLength);

extern "C" void receiveSharedOptimizationData_()
{
WSGetList(float, xI, xI_length, Real32);
int xLength;
xLength = xI_length;
receiveSharedOptimizationData(xI, xLength);
WL_RETURN_VOID();
WSReleaseList(xI, xI_length, Real32);
}

void receiveOptimizationData(int const partition, int const * const sparseDerivativeZtoYIndicesI, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndicesI, size_t const xIndicesLength, int const * const yIndicesI, size_t const yIndicesLength);

extern "C" void receiveOptimizationData_()
{
WSGet(int, partition, Integer32);
WSGetList(int, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSGetList(int, xIndicesI, xIndicesI_length, Integer32);
WSGetList(int, yIndicesI, yIndicesI_length, Integer32);
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndicesI_length;
int xIndicesLength;
xIndicesLength = xIndicesI_length;
int yIndicesLength;
yIndicesLength = yIndicesI_length;
receiveOptimizationData(partition, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength, xIndicesI, xIndicesLength, yIndicesI, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSReleaseList(xIndicesI, xIndicesI_length, Integer32);
WSReleaseList(yIndicesI, yIndicesI_length, Integer32);
}

__host__ __device__ void f(real const * const input, real * const out);

extern "C" void f_()
{
WSGetList(float, input, input_length, Real32);
float * out;
out = tmalloc<float>(lengthfz);
f(input, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(out);
}

__global__ void KERNEL_f(real const * const input, real * const out)
{
f(input, out);
}

extern "C" void f_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, input, input_length, Real32);
float * input_in;
input_in = mallocmemcpy(input, input_length);
float * out;
out = tmalloc<float>(lengthfz);
CUDAKERNEL_LAUNCH(KERNEL_f, gridDim, blockDim, input_in, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(input_in);
memoryFree(out);
}

__host__ __device__ void df(int const i, real const * const input, real * const out);

extern "C" void df_()
{
WSGet(int, i, Integer32);
WSGetList(float, input, input_length, Real32);
float * out;
out = tmalloc<float>(lengthfz);
df(i, input, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(out);
}

__global__ void KERNEL_df(int const i, real const * const input, real * const out)
{
df(i, input, out);
}

extern "C" void df_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, i, Integer32);
WSGetList(float, input, input_length, Real32);
float * input_in;
input_in = mallocmemcpy(input, input_length);
float * out;
out = tmalloc<float>(lengthfz);
CUDAKERNEL_LAUNCH(KERNEL_df, gridDim, blockDim, i, input_in, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(input_in);
memoryFree(out);
}

__host__ __device__ int cs_cumsum(int * p, int * c, int const n);

extern "C" void cs_cumsum_()
{
WSGetList(int, p, p_length, Integer32);
WSGetList(int, c, c_length, Integer32);
WSGet(int, n, Integer32);
int _result_;
_result_ = cs_cumsum(p, c, n);
WSPut(Integer32, _result_);
WSReleaseList(p, p_length, Integer32);
WSReleaseList(c, c_length, Integer32);
}

__global__ void KERNEL_cs_cumsum(int * p, int * c, int const n, int * _returns_)
{
_returns_[linear_global_threadId()] = cs_cumsum(p, c, n);
}

extern "C" void cs_cumsum_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, p, p_length, Integer32);
WSGetList(int, c, c_length, Integer32);
WSGet(int, n, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_cs_cumsum, gridDim, blockDim, p, c, n, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
WSReleaseList(p, p_length, Integer32);
WSReleaseList(c, c_length, Integer32);
memoryFree(_returns_);
}

__host__ __device__ void print(char const * const x);

extern "C" void print_()
{
WSGetCString(x);
print(x);
WL_RETURN_VOID();
WSReleaseCString(x);
}

__global__ void KERNEL_print(char const * const x)
{
print(x);
}

extern "C" void print_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetCString(x);
CUDAKERNEL_LAUNCH(KERNEL_print, gridDim, blockDim, x);
WL_RETURN_VOID();
WSReleaseCString(x);
}

__host__ __device__ void printd(int const * v, size_t n);

extern "C" void printd_()
{
WSGetList(int, v, v_length, Integer32);
int n;
n = v_length;
printd(v, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
}

__global__ void KERNEL_printd(int const * v, size_t n)
{
printd(v, n);
}

extern "C" void printd_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, v, v_length, Integer32);
int * v_in;
v_in = mallocmemcpy(v, v_length);
int n;
n = v_length;
CUDAKERNEL_LAUNCH(KERNEL_printd, gridDim, blockDim, v_in, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
memoryFree(v_in);
}

__host__ __device__ void printv(real const * v, size_t n);

extern "C" void printv_()
{
WSGetList(float, v, v_length, Real32);
int n;
n = v_length;
printv(v, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Real32);
}

__global__ void KERNEL_printv(real const * v, size_t n)
{
printv(v, n);
}

extern "C" void printv_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, v, v_length, Real32);
float * v_in;
v_in = mallocmemcpy(v, v_length);
int n;
n = v_length;
CUDAKERNEL_LAUNCH(KERNEL_printv, gridDim, blockDim, v_in, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Real32);
memoryFree(v_in);
}

__host__ __device__ void assertFinite(real const * const x, int const n);

extern "C" void assertFinite_()
{
WSGetList(float, x, x_length, Real32);
int n;
n = x_length;
assertFinite(x, n);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
}

__global__ void KERNEL_assertFinite(real const * const x, int const n)
{
assertFinite(x, n);
}

extern "C" void assertFinite_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, x, x_length, Real32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int n;
n = x_length;
CUDAKERNEL_LAUNCH(KERNEL_assertFinite, gridDim, blockDim, x_in, n);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
memoryFree(x_in);
}

__host__ __device__ int lengthzGet();

extern "C" void lengthzGet_()
{
int _result_;
_result_ = lengthzGet();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_lengthzGet(int * _returns_)
{
_returns_[linear_global_threadId()] = lengthzGet();
}

extern "C" void lengthzGet_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_lengthzGet, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int lengthfzGet();

extern "C" void lengthfzGet_()
{
int _result_;
_result_ = lengthfzGet();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_lengthfzGet(int * _returns_)
{
_returns_[linear_global_threadId()] = lengthfzGet();
}

extern "C" void lengthfzGet_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_lengthfzGet, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ void assertEachInRange(int const * v, size_t len, int const min, int const max);

extern "C" void assertEachInRange_()
{
WSGetList(int, v, v_length, Integer32);
WSGet(int, min, Integer32);
WSGet(int, max, Integer32);
int len;
len = v_length;
assertEachInRange(v, len, min, max);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
}

__global__ void KERNEL_assertEachInRange(int const * v, size_t len, int const min, int const max)
{
assertEachInRange(v, len, min, max);
}

extern "C" void assertEachInRange_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, v, v_length, Integer32);
WSGet(int, min, Integer32);
WSGet(int, max, Integer32);
int * v_in;
v_in = mallocmemcpy(v, v_length);
int len;
len = v_length;
CUDAKERNEL_LAUNCH(KERNEL_assertEachInRange, gridDim, blockDim, v_in, len, min, max);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
memoryFree(v_in);
}

__host__ __device__ void axpyWithReindexing(real * const targetBase, size_t const targetLength, real const a, real const * const addedValues, int const * const targetIndices, size_t const targetIndicesAndAddedValuesLength);

extern "C" void axpyWithReindexing_()
{
WSGetList(float, targetBase, targetBase_length, Real32);
WSGet(float, a, Real32);
WSGetList(float, addedValues, addedValues_length, Real32);
WSGetList(int, targetIndices, targetIndices_length, Integer32);
int targetLength;
targetLength = targetBase_length;
int targetIndicesAndAddedValuesLength;
targetIndicesAndAddedValuesLength = addedValues_length;
checkAllEqual(targetIndices_length, addedValues_length);
float * targetBase_inout;
targetBase_inout = mallocmemcpy(targetBase, targetLength);
axpyWithReindexing(targetBase_inout, targetLength, a, addedValues, targetIndices, targetIndicesAndAddedValuesLength);
WSPutList(Real32, targetBase_inout, targetLength);
WSReleaseList(targetBase, targetBase_length, Real32);
WSReleaseList(addedValues, addedValues_length, Real32);
WSReleaseList(targetIndices, targetIndices_length, Integer32);
memoryFree(targetBase_inout);
}

__global__ void KERNEL_axpyWithReindexing(real * const targetBase_inout, size_t const targetLength, real const a, real const * const addedValues, int const * const targetIndices, size_t const targetIndicesAndAddedValuesLength)
{
axpyWithReindexing(targetBase_inout, targetLength, a, addedValues, targetIndices, targetIndicesAndAddedValuesLength);
}

extern "C" void axpyWithReindexing_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, targetBase, targetBase_length, Real32);
WSGet(float, a, Real32);
WSGetList(float, addedValues, addedValues_length, Real32);
WSGetList(int, targetIndices, targetIndices_length, Integer32);
float * addedValues_in;
addedValues_in = mallocmemcpy(addedValues, addedValues_length);
int * targetIndices_in;
targetIndices_in = mallocmemcpy(targetIndices, targetIndices_length);
int targetLength;
targetLength = targetBase_length;
int targetIndicesAndAddedValuesLength;
targetIndicesAndAddedValuesLength = addedValues_length;
checkAllEqual(targetIndices_length, addedValues_length);
float * targetBase_inout;
targetBase_inout = mallocmemcpy(targetBase, targetLength);
CUDAKERNEL_LAUNCH(KERNEL_axpyWithReindexing, gridDim, blockDim, targetBase_inout, targetLength, a, addedValues_in, targetIndices_in, targetIndicesAndAddedValuesLength);
WSPutList(Real32, targetBase_inout, targetLength);
WSReleaseList(targetBase, targetBase_length, Real32);
WSReleaseList(addedValues, addedValues_length, Real32);
WSReleaseList(targetIndices, targetIndices_length, Integer32);
memoryFree(addedValues_in);
memoryFree(targetIndices_in);
memoryFree(targetBase_inout);
}

__host__ __device__ void extract(real * const target, real const * const source, size_t const sourceLength, int const * const sourceIndices, size_t const sourceIndicesAndTargetLength);

extern "C" void extract_()
{
WSGetList(float, source, source_length, Real32);
WSGetList(int, sourceIndices, sourceIndices_length, Integer32);
int sourceLength;
sourceLength = source_length;
int sourceIndicesAndTargetLength;
sourceIndicesAndTargetLength = sourceIndices_length;
float * target;
target = tmalloc<float>(sourceIndicesAndTargetLength);
extract(target, source, sourceLength, sourceIndices, sourceIndicesAndTargetLength);
WSPutList(Real32, target, sourceIndicesAndTargetLength);
WSReleaseList(source, source_length, Real32);
WSReleaseList(sourceIndices, sourceIndices_length, Integer32);
memoryFree(target);
}

__global__ void KERNEL_extract(real * const target, real const * const source, size_t const sourceLength, int const * const sourceIndices, size_t const sourceIndicesAndTargetLength)
{
extract(target, source, sourceLength, sourceIndices, sourceIndicesAndTargetLength);
}

extern "C" void extract_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, source, source_length, Real32);
WSGetList(int, sourceIndices, sourceIndices_length, Integer32);
float * source_in;
source_in = mallocmemcpy(source, source_length);
int * sourceIndices_in;
sourceIndices_in = mallocmemcpy(sourceIndices, sourceIndices_length);
int sourceLength;
sourceLength = source_length;
int sourceIndicesAndTargetLength;
sourceIndicesAndTargetLength = sourceIndices_length;
float * target;
target = tmalloc<float>(sourceIndicesAndTargetLength);
CUDAKERNEL_LAUNCH(KERNEL_extract, gridDim, blockDim, target, source_in, sourceLength, sourceIndices_in, sourceIndicesAndTargetLength);
WSPutList(Real32, target, sourceIndicesAndTargetLength);
WSReleaseList(source, source_length, Real32);
WSReleaseList(sourceIndices, sourceIndices_length, Integer32);
memoryFree(source_in);
memoryFree(sourceIndices_in);
memoryFree(target);
}

__host__ __device__ void getY(int partition, real * const outY, int lengthY);

extern "C" void getY_()
{
WSGet(int, partition, Integer32);
WSGet(int, lengthY, Integer32);
float * outY;
outY = tmalloc<float>(lengthY);
getY(partition, outY, lengthY);
WSPutList(Real32, outY, lengthY);
memoryFree(outY);
}

__global__ void KERNEL_getY(int partition, real * const outY, int lengthY)
{
getY(partition, outY, lengthY);
}

extern "C" void getY_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, partition, Integer32);
WSGet(int, lengthY, Integer32);
float * outY;
outY = tmalloc<float>(lengthY);
CUDAKERNEL_LAUNCH(KERNEL_getY, gridDim, blockDim, partition, outY, lengthY);
WSPutList(Real32, outY, lengthY);
memoryFree(outY);
}

__host__ __device__ void buildFxAndJFxAndSolveRepeatedly(int const partition, int const iterations);

extern "C" void buildFxAndJFxAndSolveRepeatedly_()
{
WSGet(int, partition, Integer32);
WSGet(int, iterations, Integer32);
buildFxAndJFxAndSolveRepeatedly(partition, iterations);
WL_RETURN_VOID();
}

__global__ void KERNEL_buildFxAndJFxAndSolveRepeatedly(int const partition, int const iterations)
{
buildFxAndJFxAndSolveRepeatedly(partition, iterations);
}

extern "C" void buildFxAndJFxAndSolveRepeatedly_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, partition, Integer32);
WSGet(int, iterations, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_buildFxAndJFxAndSolveRepeatedly, gridDim, blockDim, partition, iterations);
WL_RETURN_VOID();
}

__host__ __device__ void buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(int const iterations);

extern "C" void buildFxAndJFxAndSolveRepeatedlyThreadIdPartition_()
{
WSGet(int, iterations, Integer32);
buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(iterations);
WL_RETURN_VOID();
}

__global__ void KERNEL_buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(int const iterations)
{
buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(iterations);
}

extern "C" void buildFxAndJFxAndSolveRepeatedlyThreadIdPartition_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, iterations, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_buildFxAndJFxAndSolveRepeatedlyThreadIdPartition, gridDim, blockDim, iterations);
WL_RETURN_VOID();
}

__host__ __device__ void receiveAndPrintOptimizationData(int const lengthz, int const lengthfz, real const * const x, size_t const xLength, int const * const sparseDerivativeZtoYIndices, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndices, size_t const xIndicesLength, int const * const yIndices, size_t const yIndicesLength);

extern "C" void receiveAndPrintOptimizationData_()
{
WSGet(int, lengthz, Integer32);
WSGet(int, lengthfz, Integer32);
WSGetList(float, x, x_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSGetList(int, xIndices, xIndices_length, Integer32);
WSGetList(int, yIndices, yIndices_length, Integer32);
int xLength;
xLength = x_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndices_length;
int xIndicesLength;
xIndicesLength = xIndices_length;
int yIndicesLength;
yIndicesLength = yIndices_length;
receiveAndPrintOptimizationData(lengthz, lengthfz, x, xLength, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndicesLength, xIndices, xIndicesLength, yIndices, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSReleaseList(xIndices, xIndices_length, Integer32);
WSReleaseList(yIndices, yIndices_length, Integer32);
}

__global__ void KERNEL_receiveAndPrintOptimizationData(int const lengthz, int const lengthfz, real const * const x, size_t const xLength, int const * const sparseDerivativeZtoYIndices, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndices, size_t const xIndicesLength, int const * const yIndices, size_t const yIndicesLength)
{
receiveAndPrintOptimizationData(lengthz, lengthfz, x, xLength, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndicesLength, xIndices, xIndicesLength, yIndices, yIndicesLength);
}

extern "C" void receiveAndPrintOptimizationData_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, lengthz, Integer32);
WSGet(int, lengthfz, Integer32);
WSGetList(float, x, x_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSGetList(int, xIndices, xIndices_length, Integer32);
WSGetList(int, yIndices, yIndices_length, Integer32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int * sparseDerivativeZtoYIndices_in;
sparseDerivativeZtoYIndices_in = mallocmemcpy(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length);
int * xIndices_in;
xIndices_in = mallocmemcpy(xIndices, xIndices_length);
int * yIndices_in;
yIndices_in = mallocmemcpy(yIndices, yIndices_length);
int xLength;
xLength = x_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndices_length;
int xIndicesLength;
xIndicesLength = xIndices_length;
int yIndicesLength;
yIndicesLength = yIndices_length;
CUDAKERNEL_LAUNCH(KERNEL_receiveAndPrintOptimizationData, gridDim, blockDim, lengthz, lengthfz, x_in, xLength, sparseDerivativeZtoYIndices_in, sparseDerivativeZtoYIndicesLength, xIndices_in, xIndicesLength, yIndices_in, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSReleaseList(xIndices, xIndices_length, Integer32);
WSReleaseList(yIndices, yIndices_length, Integer32);
memoryFree(x_in);
memoryFree(sparseDerivativeZtoYIndices_in);
memoryFree(xIndices_in);
memoryFree(yIndices_in);
}

__host__ __device__ void makeAndPrintSparseMatrix(size_t const m, size_t const n, real * x, size_t xlen, int * ij, size_t const ijlen);

extern "C" void makeAndPrintSparseMatrix_()
{
WSGet(long long, m, Integer64);
WSGet(long long, n, Integer64);
WSGetList(float, x, x_length, Real32);
WSGetList(int, ij, ij_length, Integer32);
int xlen;
xlen = x_length;
int ijlen;
ijlen = ij_length;
makeAndPrintSparseMatrix(m, n, x, xlen, ij, ijlen);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(ij, ij_length, Integer32);
}

__global__ void KERNEL_makeAndPrintSparseMatrix(size_t const m, size_t const n, real * x, size_t xlen, int * ij, size_t const ijlen)
{
makeAndPrintSparseMatrix(m, n, x, xlen, ij, ijlen);
}

extern "C" void makeAndPrintSparseMatrix_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, m, Integer64);
WSGet(long long, n, Integer64);
WSGetList(float, x, x_length, Real32);
WSGetList(int, ij, ij_length, Integer32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int * ij_in;
ij_in = mallocmemcpy(ij, ij_length);
int xlen;
xlen = x_length;
int ijlen;
ijlen = ij_length;
CUDAKERNEL_LAUNCH(KERNEL_makeAndPrintSparseMatrix, gridDim, blockDim, m, n, x_in, xlen, ij_in, ijlen);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(ij, ij_length, Integer32);
memoryFree(x_in);
memoryFree(ij_in);
}

__host__ __device__ void testMain();

extern "C" void testMain_()
{
testMain();
WL_RETURN_VOID();
}

__global__ void KERNEL_testMain()
{
testMain();
}

extern "C" void testMain_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_testMain, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ void xx43();

extern "C" void xx43_()
{
xx43();
WL_RETURN_VOID();
}

__global__ void KERNEL_xx43()
{
xx43();
}

extern "C" void xx43_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_xx43, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ int get42();

extern "C" void get42_()
{
int _result_;
_result_ = get42();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_get42(int * _returns_)
{
_returns_[linear_global_threadId()] = get42();
}

extern "C" void get42_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_get42, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int add(int x, int y);

extern "C" void add_()
{
WSGet(int, x, Integer32);
WSGet(int, y, Integer32);
int _result_;
_result_ = add(x, y);
WSPut(Integer32, _result_);
}

__global__ void KERNEL_add(int x, int y, int * _returns_)
{
_returns_[linear_global_threadId()] = add(x, y);
}

extern "C" void add_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, x, Integer32);
WSGet(int, y, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_add, gridDim, blockDim, x, y, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ real addf(real x, real y);

extern "C" void addf_()
{
WSGet(float, x, Real32);
WSGet(float, y, Real32);
real _result_;
_result_ = addf(x, y);
WSPut(Real32, _result_);
}

__global__ void KERNEL_addf(real x, real y, float * _returns_)
{
_returns_[linear_global_threadId()] = addf(x, y);
}

extern "C" void addf_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(float, x, Real32);
WSGet(float, y, Real32);
float * _returns_;
_returns_ = tmalloc<float>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_addf, gridDim, blockDim, x, y, _returns_);
WSPutList(Real32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int multiout(int * x, int l);

extern "C" void multiout_()
{
WSGetList(int, x, x_length, Integer32);
int l;
l = x_length;
int * x_inout;
x_inout = mallocmemcpy(x, l);
int _result_;
_result_ = multiout(x_inout, l);
WSPutFunction(stdlink, "Association", 2);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("return");
WSPut(Integer32, _result_);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("x");
WSPutList(Integer32, x_inout, l);
WSReleaseList(x, x_length, Integer32);
memoryFree(x_inout);
}

__global__ void KERNEL_multiout(int * x_inout, int l, int * _returns_)
{
_returns_[linear_global_threadId()] = multiout(x_inout, l);
}

extern "C" void multiout_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, x, x_length, Integer32);
int l;
l = x_length;
int * x_inout;
x_inout = mallocmemcpy(x, l);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_multiout, gridDim, blockDim, x_inout, l, _returns_);
WSPutFunction(stdlink, "Association", 2);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("x");
WSPutList(Integer32, x_inout, l);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("returns");
WSPutList(Integer32, _returns_, gridDim * blockDim);
WSReleaseList(x, x_length, Integer32);
memoryFree(x_inout);
memoryFree(_returns_);
}

extern int dprintEnabled;

extern "C" void dprintEnabled_get()
{
WSPut(Integer32, dprintEnabled);
}

extern int xx;

extern "C" void xx_get()
{
WSPut(Integer32, xx);
}

extern real a;

extern "C" void a_get()
{
WSPut(Real32, a);
}

extern real b;

extern "C" void b_get()
{
WSPut(Real32, b);
}

extern real yy;

extern "C" void yy_get()
{
WSPut(Real32, yy);
}

extern real * x;

extern int lengthx;

extern "C" void x_get()
{
WSPutList(Real32, x, lengthx);
}

extern "C" void dprintEnabled_set()
{
WSGetInteger32(stdlink,&dprintEnabled);
WL_RETURN_VOID();
}

extern "C" void xx_set()
{
WSGetInteger32(stdlink,&xx);
WL_RETURN_VOID();
}

extern "C" void a_set()
{
WSGetReal32(stdlink,&a);
WL_RETURN_VOID();
}

extern "C" void b_set()
{
WSGetReal32(stdlink,&b);
WL_RETURN_VOID();
}

extern "C" void yy_set()
{
WSGetReal32(stdlink,&yy);
WL_RETURN_VOID();
}

extern "C" void x_set()
{
WSGetList(float, x_in, x_in_length, Real32);
freemalloctmemcpy(&x, x_in, x_in_length);
lengthx = x_in_length;
WL_RETURN_VOID();
WSReleaseList(x_in, x_in_length, Real32);
}


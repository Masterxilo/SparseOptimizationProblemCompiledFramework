#include "hip/hip_runtime.h"
void setPartitions(size_t newPartitionsCount);

extern "C" void setPartitions_()
{
WSGet(long long, newPartitionsCount, Integer64);
setPartitions(newPartitionsCount);
WL_RETURN_VOID();
}

void receiveSharedOptimizationData(float const * const xI, size_t const xLength);

extern "C" void receiveSharedOptimizationData_()
{
WSGetList(float, xI, xI_length, Real32);
int xLength;
xLength = xI_length;
receiveSharedOptimizationData(xI, xLength);
WL_RETURN_VOID();
WSReleaseList(xI, xI_length, Real32);
}

void receiveOptimizationData(int const partition, int const * const sparseDerivativeZtoYIndicesI, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndicesI, size_t const xIndicesLength, int const * const yIndicesI, size_t const yIndicesLength);

extern "C" void receiveOptimizationData_()
{
WSGet(int, partition, Integer32);
WSGetList(int, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSGetList(int, xIndicesI, xIndicesI_length, Integer32);
WSGetList(int, yIndicesI, yIndicesI_length, Integer32);
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndicesI_length;
int xIndicesLength;
xIndicesLength = xIndicesI_length;
int yIndicesLength;
yIndicesLength = yIndicesI_length;
receiveOptimizationData(partition, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength, xIndicesI, xIndicesLength, yIndicesI, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSReleaseList(xIndicesI, xIndicesI_length, Integer32);
WSReleaseList(yIndicesI, yIndicesI_length, Integer32);
}

__host__ __device__ void f(float const * const input, float * const out);

extern "C" void f_()
{
WSGetList(float, input, input_length, Real32);
float * out;
out = tmalloc<float>(lengthfz);
f(input, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(out);
}

__global__ void KERNEL_f(float const * const input, float * const out)
{
f(input, out);
}

extern "C" void f_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, input, input_length, Real32);
float * input_in;
input_in = mallocmemcpy(input, input_length);
float * out;
out = tmalloc<float>(lengthfz);
CUDAKERNEL_LAUNCH(KERNEL_f, gridDim, blockDim, input_in, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(input_in);
memoryFree(out);
}

__host__ __device__ void df(unsigned int const i, float const * const input, float * const out);

extern "C" void df_()
{
WSGet(int, i, Integer32);
WSGetList(float, input, input_length, Real32);
float * out;
out = tmalloc<float>(lengthfz);
df(i, input, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(out);
}

__global__ void KERNEL_df(unsigned int const i, float const * const input, float * const out)
{
df(i, input, out);
}

extern "C" void df_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, i, Integer32);
WSGetList(float, input, input_length, Real32);
float * input_in;
input_in = mallocmemcpy(input, input_length);
float * out;
out = tmalloc<float>(lengthfz);
CUDAKERNEL_LAUNCH(KERNEL_df, gridDim, blockDim, i, input_in, out);
WSPutList(Real32, out, lengthfz);
WSReleaseList(input, input_length, Real32);
memoryFree(input_in);
memoryFree(out);
}

__host__ __device__ void print(char const * const x);

extern "C" void print_()
{
WSGetCString(x);
print(x);
WL_RETURN_VOID();
WSReleaseCString(x);
}

__global__ void KERNEL_print(char const * const x)
{
print(x);
}

extern "C" void print_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetCString(x);
CUDAKERNEL_LAUNCH(KERNEL_print, gridDim, blockDim, x);
WL_RETURN_VOID();
WSReleaseCString(x);
}

__host__ __device__ void printd(int const * v, size_t n);

extern "C" void printd_()
{
WSGetList(int, v, v_length, Integer32);
int n;
n = v_length;
printd(v, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
}

__global__ void KERNEL_printd(int const * v, size_t n)
{
printd(v, n);
}

extern "C" void printd_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, v, v_length, Integer32);
int * v_in;
v_in = mallocmemcpy(v, v_length);
int n;
n = v_length;
CUDAKERNEL_LAUNCH(KERNEL_printd, gridDim, blockDim, v_in, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
memoryFree(v_in);
}

__host__ __device__ void printv(float const * v, size_t n);

extern "C" void printv_()
{
WSGetList(float, v, v_length, Real32);
int n;
n = v_length;
printv(v, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Real32);
}

__global__ void KERNEL_printv(float const * v, size_t n)
{
printv(v, n);
}

extern "C" void printv_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, v, v_length, Real32);
float * v_in;
v_in = mallocmemcpy(v, v_length);
int n;
n = v_length;
CUDAKERNEL_LAUNCH(KERNEL_printv, gridDim, blockDim, v_in, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Real32);
memoryFree(v_in);
}

__host__ __device__ int lengthzGet();

extern "C" void lengthzGet_()
{
int _result_;
_result_ = lengthzGet();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_lengthzGet(int * _returns_)
{
_returns_[linear_global_threadId()] = lengthzGet();
}

extern "C" void lengthzGet_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_lengthzGet, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int lengthfzGet();

extern "C" void lengthfzGet_()
{
int _result_;
_result_ = lengthfzGet();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_lengthfzGet(int * _returns_)
{
_returns_[linear_global_threadId()] = lengthfzGet();
}

extern "C" void lengthfzGet_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_lengthfzGet, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ void getY(int partition, float * const outY, int lengthY);

extern "C" void getY_()
{
WSGet(int, partition, Integer32);
WSGet(int, lengthY, Integer32);
float * outY;
outY = tmalloc<float>(lengthY);
getY(partition, outY, lengthY);
WSPutList(Real32, outY, lengthY);
memoryFree(outY);
}

__global__ void KERNEL_getY(int partition, float * const outY, int lengthY)
{
getY(partition, outY, lengthY);
}

extern "C" void getY_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, partition, Integer32);
WSGet(int, lengthY, Integer32);
float * outY;
outY = tmalloc<float>(lengthY);
CUDAKERNEL_LAUNCH(KERNEL_getY, gridDim, blockDim, partition, outY, lengthY);
WSPutList(Real32, outY, lengthY);
memoryFree(outY);
}

__host__ __device__ void buildFxAndJFxAndSolveRepeatedly(int const partition, int const iterations);

extern "C" void buildFxAndJFxAndSolveRepeatedly_()
{
WSGet(int, partition, Integer32);
WSGet(int, iterations, Integer32);
buildFxAndJFxAndSolveRepeatedly(partition, iterations);
WL_RETURN_VOID();
}

__global__ void KERNEL_buildFxAndJFxAndSolveRepeatedly(int const partition, int const iterations)
{
buildFxAndJFxAndSolveRepeatedly(partition, iterations);
}

extern "C" void buildFxAndJFxAndSolveRepeatedly_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, partition, Integer32);
WSGet(int, iterations, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_buildFxAndJFxAndSolveRepeatedly, gridDim, blockDim, partition, iterations);
WL_RETURN_VOID();
}

__host__ __device__ void buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(int const iterations);

extern "C" void buildFxAndJFxAndSolveRepeatedlyThreadIdPartition_()
{
WSGet(int, iterations, Integer32);
buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(iterations);
WL_RETURN_VOID();
}

__global__ void KERNEL_buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(int const iterations)
{
buildFxAndJFxAndSolveRepeatedlyThreadIdPartition(iterations);
}

extern "C" void buildFxAndJFxAndSolveRepeatedlyThreadIdPartition_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, iterations, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_buildFxAndJFxAndSolveRepeatedlyThreadIdPartition, gridDim, blockDim, iterations);
WL_RETURN_VOID();
}

__host__ __device__ void receiveAndPrintOptimizationData(int const lengthz, int const lengthfz, float const * const x, size_t const xLength, int const * const sparseDerivativeZtoYIndices, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndices, size_t const xIndicesLength, int const * const yIndices, size_t const yIndicesLength);

extern "C" void receiveAndPrintOptimizationData_()
{
WSGet(int, lengthz, Integer32);
WSGet(int, lengthfz, Integer32);
WSGetList(float, x, x_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSGetList(int, xIndices, xIndices_length, Integer32);
WSGetList(int, yIndices, yIndices_length, Integer32);
int xLength;
xLength = x_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndices_length;
int xIndicesLength;
xIndicesLength = xIndices_length;
int yIndicesLength;
yIndicesLength = yIndices_length;
receiveAndPrintOptimizationData(lengthz, lengthfz, x, xLength, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndicesLength, xIndices, xIndicesLength, yIndices, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSReleaseList(xIndices, xIndices_length, Integer32);
WSReleaseList(yIndices, yIndices_length, Integer32);
}

__global__ void KERNEL_receiveAndPrintOptimizationData(int const lengthz, int const lengthfz, float const * const x, size_t const xLength, int const * const sparseDerivativeZtoYIndices, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndices, size_t const xIndicesLength, int const * const yIndices, size_t const yIndicesLength)
{
receiveAndPrintOptimizationData(lengthz, lengthfz, x, xLength, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndicesLength, xIndices, xIndicesLength, yIndices, yIndicesLength);
}

extern "C" void receiveAndPrintOptimizationData_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, lengthz, Integer32);
WSGet(int, lengthfz, Integer32);
WSGetList(float, x, x_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSGetList(int, xIndices, xIndices_length, Integer32);
WSGetList(int, yIndices, yIndices_length, Integer32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int * sparseDerivativeZtoYIndices_in;
sparseDerivativeZtoYIndices_in = mallocmemcpy(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length);
int * xIndices_in;
xIndices_in = mallocmemcpy(xIndices, xIndices_length);
int * yIndices_in;
yIndices_in = mallocmemcpy(yIndices, yIndices_length);
int xLength;
xLength = x_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndices_length;
int xIndicesLength;
xIndicesLength = xIndices_length;
int yIndicesLength;
yIndicesLength = yIndices_length;
CUDAKERNEL_LAUNCH(KERNEL_receiveAndPrintOptimizationData, gridDim, blockDim, lengthz, lengthfz, x_in, xLength, sparseDerivativeZtoYIndices_in, sparseDerivativeZtoYIndicesLength, xIndices_in, xIndicesLength, yIndices_in, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSReleaseList(xIndices, xIndices_length, Integer32);
WSReleaseList(yIndices, yIndices_length, Integer32);
memoryFree(x_in);
memoryFree(sparseDerivativeZtoYIndices_in);
memoryFree(xIndices_in);
memoryFree(yIndices_in);
}

__host__ __device__ void makeAndPrintSparseMatrix(size_t const m, size_t const n, float * x, size_t xlen, int * ij, size_t const ijlen);

extern "C" void makeAndPrintSparseMatrix_()
{
WSGet(long long, m, Integer64);
WSGet(long long, n, Integer64);
WSGetList(float, x, x_length, Real32);
WSGetList(int, ij, ij_length, Integer32);
int xlen;
xlen = x_length;
int ijlen;
ijlen = ij_length;
makeAndPrintSparseMatrix(m, n, x, xlen, ij, ijlen);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(ij, ij_length, Integer32);
}

__global__ void KERNEL_makeAndPrintSparseMatrix(size_t const m, size_t const n, float * x, size_t xlen, int * ij, size_t const ijlen)
{
makeAndPrintSparseMatrix(m, n, x, xlen, ij, ijlen);
}

extern "C" void makeAndPrintSparseMatrix_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, m, Integer64);
WSGet(long long, n, Integer64);
WSGetList(float, x, x_length, Real32);
WSGetList(int, ij, ij_length, Integer32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int * ij_in;
ij_in = mallocmemcpy(ij, ij_length);
int xlen;
xlen = x_length;
int ijlen;
ijlen = ij_length;
CUDAKERNEL_LAUNCH(KERNEL_makeAndPrintSparseMatrix, gridDim, blockDim, m, n, x_in, xlen, ij_in, ijlen);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(ij, ij_length, Integer32);
memoryFree(x_in);
memoryFree(ij_in);
}

__host__ __device__ void testMain();

extern "C" void testMain_()
{
testMain();
WL_RETURN_VOID();
}

__global__ void KERNEL_testMain()
{
testMain();
}

extern "C" void testMain_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_testMain, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ void xx43();

extern "C" void xx43_()
{
xx43();
WL_RETURN_VOID();
}

__global__ void KERNEL_xx43()
{
xx43();
}

extern "C" void xx43_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_xx43, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ int get42();

extern "C" void get42_()
{
int _result_;
_result_ = get42();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_get42(int * _returns_)
{
_returns_[linear_global_threadId()] = get42();
}

extern "C" void get42_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_get42, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int add(int x, int y);

extern "C" void add_()
{
WSGet(int, x, Integer32);
WSGet(int, y, Integer32);
int _result_;
_result_ = add(x, y);
WSPut(Integer32, _result_);
}

__global__ void KERNEL_add(int x, int y, int * _returns_)
{
_returns_[linear_global_threadId()] = add(x, y);
}

extern "C" void add_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, x, Integer32);
WSGet(int, y, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_add, gridDim, blockDim, x, y, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ float addf(float x, float y);

extern "C" void addf_()
{
WSGet(float, x, Real32);
WSGet(float, y, Real32);
float _result_;
_result_ = addf(x, y);
WSPut(Real32, _result_);
}

__global__ void KERNEL_addf(float x, float y, float * _returns_)
{
_returns_[linear_global_threadId()] = addf(x, y);
}

extern "C" void addf_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(float, x, Real32);
WSGet(float, y, Real32);
float * _returns_;
_returns_ = tmalloc<float>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_addf, gridDim, blockDim, x, y, _returns_);
WSPutList(Real32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int multiout(int * x, int l);

extern "C" void multiout_()
{
WSGetList(int, x, x_length, Integer32);
int l;
l = x_length;
int * x_inout;
x_inout = mallocmemcpy(x, l);
int _result_;
_result_ = multiout(x_inout, l);
WSPutFunction(stdlink, "Association", 2);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("return");
WSPut(Integer32, _result_);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("x");
WSPutList(Integer32, x_inout, l);
WSReleaseList(x, x_length, Integer32);
memoryFree(x_inout);
}

__global__ void KERNEL_multiout(int * x_inout, int l, int * _returns_)
{
_returns_[linear_global_threadId()] = multiout(x_inout, l);
}

extern "C" void multiout_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, x, x_length, Integer32);
int l;
l = x_length;
int * x_inout;
x_inout = mallocmemcpy(x, l);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_multiout, gridDim, blockDim, x_inout, l, _returns_);
WSPutFunction(stdlink, "Association", 2);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("x");
WSPutList(Integer32, x_inout, l);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("returns");
WSPutList(Integer32, _returns_, gridDim * blockDim);
WSReleaseList(x, x_length, Integer32);
memoryFree(x_inout);
memoryFree(_returns_);
}

__host__ __device__ void mainc();

extern "C" void mainc_()
{
mainc();
WL_RETURN_VOID();
}

__global__ void KERNEL_mainc()
{
mainc();
}

extern "C" void mainc_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_mainc, gridDim, blockDim);
WL_RETURN_VOID();
}

extern int dprintEnabled;

extern "C" void dprintEnabled_get()
{
WSPut(Integer32, dprintEnabled);
}

extern int xx;

extern "C" void xx_get()
{
WSPut(Integer32, xx);
}

extern float a;

extern "C" void a_get()
{
WSPut(Real32, a);
}

extern float b;

extern "C" void b_get()
{
WSPut(Real32, b);
}

extern float yy;

extern "C" void yy_get()
{
WSPut(Real32, yy);
}

extern float * x;

extern int lengthx;

extern "C" void x_get()
{
WSPutList(Real32, x, lengthx);
}

extern "C" void dprintEnabled_set()
{
WSGetInteger32(stdlink,&dprintEnabled);
WL_RETURN_VOID();
}

extern "C" void xx_set()
{
WSGetInteger32(stdlink,&xx);
WL_RETURN_VOID();
}

extern "C" void a_set()
{
WSGetReal32(stdlink,&a);
WL_RETURN_VOID();
}

extern "C" void b_set()
{
WSGetReal32(stdlink,&b);
WL_RETURN_VOID();
}

extern "C" void yy_set()
{
WSGetReal32(stdlink,&yy);
WL_RETURN_VOID();
}

extern "C" void x_set()
{
WSGetList(float, x_in, x_in_length, Real32);
freemalloctmemcpy(&x, x_in, x_in_length);
lengthx = x_in_length;
WL_RETURN_VOID();
WSReleaseList(x_in, x_in_length, Real32);
}


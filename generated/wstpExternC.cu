#include "hip/hip_runtime.h"
#include "framework.h"
void receiveOptimizationData(real const * const xI, size_t const xLength, int const * const sparseDerivativeZtoYIndicesI, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndicesI, size_t const xIndicesLength, int const * const yIndicesI, size_t const yIndicesLength);

extern "C" void receiveOptimizationData_()
{
WSGetList(float, xI, xI_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSGetList(int, xIndicesI, xIndicesI_length, Integer32);
WSGetList(int, yIndicesI, yIndicesI_length, Integer32);
int xLength;
xLength = xI_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndicesI_length;
int xIndicesLength;
xIndicesLength = xIndicesI_length;
int yIndicesLength;
yIndicesLength = yIndicesI_length;
receiveOptimizationData(xI, xLength, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength, xIndicesI, xIndicesLength, yIndicesI, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(xI, xI_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSReleaseList(xIndicesI, xIndicesI_length, Integer32);
WSReleaseList(yIndicesI, yIndicesI_length, Integer32);
}

void receiveOptimizationDataBuildFxAndJFxAndSolveRepeatedly(real const * const xI, size_t const xLength, int const * const sparseDerivativeZtoYIndicesI, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndicesI, size_t const xIndicesLength, int const * const yIndicesI, size_t const yIndicesLength, int const iterations);

extern "C" void receiveOptimizationDataBuildFxAndJFxAndSolveRepeatedly_()
{
WSGetList(float, xI, xI_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSGetList(int, xIndicesI, xIndicesI_length, Integer32);
WSGetList(int, yIndicesI, yIndicesI_length, Integer32);
WSGet(int, iterations, Integer32);
int xLength;
xLength = xI_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndicesI_length;
int xIndicesLength;
xIndicesLength = xIndicesI_length;
int yIndicesLength;
yIndicesLength = yIndicesI_length;
receiveOptimizationDataBuildFxAndJFxAndSolveRepeatedly(xI, xLength, sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesLength, xIndicesI, xIndicesLength, yIndicesI, yIndicesLength, iterations);
WL_RETURN_VOID();
WSReleaseList(xI, xI_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndicesI, sparseDerivativeZtoYIndicesI_length, Integer32);
WSReleaseList(xIndicesI, xIndicesI_length, Integer32);
WSReleaseList(yIndicesI, yIndicesI_length, Integer32);
}

__host__ __device__ int lengthzGet();

extern "C" void lengthzGet_()
{
int _result_;
_result_ = lengthzGet();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_lengthzGet(int * _returns_)
{
_returns_[linear_global_threadId()] = lengthzGet();
}

extern "C" void lengthzGet_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_lengthzGet, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int lengthfzGet();

extern "C" void lengthfzGet_()
{
int _result_;
_result_ = lengthfzGet();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_lengthfzGet(int * _returns_)
{
_returns_[linear_global_threadId()] = lengthfzGet();
}

extern "C" void lengthfzGet_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_lengthfzGet, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ void print(char const * const x);

extern "C" void print_()
{
WSGetCString(x);
print(x);
WL_RETURN_VOID();
WSReleaseCString(x);
}

__global__ void KERNEL_print(char const * const x)
{
print(x);
}

extern "C" void print_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetCString(x);
CUDAKERNEL_LAUNCH(KERNEL_print, gridDim, blockDim, x);
WL_RETURN_VOID();
WSReleaseCString(x);
}

__host__ __device__ void printv(real const * v, size_t n);

extern "C" void printv_()
{
WSGetList(float, v, v_length, Real32);
int n;
n = v_length;
printv(v, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Real32);
}

__global__ void KERNEL_printv(real const * v, size_t n)
{
printv(v, n);
}

extern "C" void printv_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, v, v_length, Real32);
float * v_in;
v_in = mallocmemcpy(v, v_length);
int n;
n = v_length;
CUDAKERNEL_LAUNCH(KERNEL_printv, gridDim, blockDim, v_in, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Real32);
memoryFree(v_in);
}

__host__ __device__ void printd(int const * v, size_t n);

extern "C" void printd_()
{
WSGetList(int, v, v_length, Integer32);
int n;
n = v_length;
printd(v, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
}

__global__ void KERNEL_printd(int const * v, size_t n)
{
printd(v, n);
}

extern "C" void printd_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, v, v_length, Integer32);
int * v_in;
v_in = mallocmemcpy(v, v_length);
int n;
n = v_length;
CUDAKERNEL_LAUNCH(KERNEL_printd, gridDim, blockDim, v_in, n);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
memoryFree(v_in);
}

__host__ __device__ void assertEachInRange(int const * v, size_t len, int const min, int const max);

extern "C" void assertEachInRange_()
{
WSGetList(int, v, v_length, Integer32);
WSGet(int, min, Integer32);
WSGet(int, max, Integer32);
int len;
len = v_length;
assertEachInRange(v, len, min, max);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
}

__global__ void KERNEL_assertEachInRange(int const * v, size_t len, int const min, int const max)
{
assertEachInRange(v, len, min, max);
}

extern "C" void assertEachInRange_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, v, v_length, Integer32);
WSGet(int, min, Integer32);
WSGet(int, max, Integer32);
int * v_in;
v_in = mallocmemcpy(v, v_length);
int len;
len = v_length;
CUDAKERNEL_LAUNCH(KERNEL_assertEachInRange, gridDim, blockDim, v_in, len, min, max);
WL_RETURN_VOID();
WSReleaseList(v, v_length, Integer32);
memoryFree(v_in);
}

__host__ __device__ void axpyWithReindexing(real * const targetBase, size_t const targetLength, real const a, real const * const addedValues, int const * const targetIndices, size_t const targetIndicesAndAddedValuesLength);

extern "C" void axpyWithReindexing_()
{
WSGetList(float, targetBase, targetBase_length, Real32);
WSGet(float, a, Real32);
WSGetList(float, addedValues, addedValues_length, Real32);
WSGetList(int, targetIndices, targetIndices_length, Integer32);
int targetLength;
targetLength = targetBase_length;
int targetIndicesAndAddedValuesLength;
targetIndicesAndAddedValuesLength = addedValues_length;
checkAllEqual(targetIndices_length, addedValues_length);
float * targetBase_inout;
targetBase_inout = mallocmemcpy(targetBase, targetLength);
axpyWithReindexing(targetBase_inout, targetLength, a, addedValues, targetIndices, targetIndicesAndAddedValuesLength);
WSPutList(Real32, targetBase_inout, targetLength);
WSReleaseList(targetBase, targetBase_length, Real32);
WSReleaseList(addedValues, addedValues_length, Real32);
WSReleaseList(targetIndices, targetIndices_length, Integer32);
memoryFree(targetBase_inout);
}

__global__ void KERNEL_axpyWithReindexing(real * const targetBase_inout, size_t const targetLength, real const a, real const * const addedValues, int const * const targetIndices, size_t const targetIndicesAndAddedValuesLength)
{
axpyWithReindexing(targetBase_inout, targetLength, a, addedValues, targetIndices, targetIndicesAndAddedValuesLength);
}

extern "C" void axpyWithReindexing_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, targetBase, targetBase_length, Real32);
WSGet(float, a, Real32);
WSGetList(float, addedValues, addedValues_length, Real32);
WSGetList(int, targetIndices, targetIndices_length, Integer32);
float * addedValues_in;
addedValues_in = mallocmemcpy(addedValues, addedValues_length);
int * targetIndices_in;
targetIndices_in = mallocmemcpy(targetIndices, targetIndices_length);
int targetLength;
targetLength = targetBase_length;
int targetIndicesAndAddedValuesLength;
targetIndicesAndAddedValuesLength = addedValues_length;
checkAllEqual(targetIndices_length, addedValues_length);
float * targetBase_inout;
targetBase_inout = mallocmemcpy(targetBase, targetLength);
CUDAKERNEL_LAUNCH(KERNEL_axpyWithReindexing, gridDim, blockDim, targetBase_inout, targetLength, a, addedValues_in, targetIndices_in, targetIndicesAndAddedValuesLength);
WSPutList(Real32, targetBase_inout, targetLength);
WSReleaseList(targetBase, targetBase_length, Real32);
WSReleaseList(addedValues, addedValues_length, Real32);
WSReleaseList(targetIndices, targetIndices_length, Integer32);
memoryFree(addedValues_in);
memoryFree(targetIndices_in);
memoryFree(targetBase_inout);
}

__host__ __device__ void extract(real * const target, real const * const source, size_t const sourceLength, int const * const sourceIndices, size_t const sourceIndicesAndTargetLength);

extern "C" void extract_()
{
WSGetList(float, source, source_length, Real32);
WSGetList(int, sourceIndices, sourceIndices_length, Integer32);
int sourceLength;
sourceLength = source_length;
int sourceIndicesAndTargetLength;
sourceIndicesAndTargetLength = sourceIndices_length;
float * target;
target = tmalloc<float>(sourceIndicesAndTargetLength);
extract(target, source, sourceLength, sourceIndices, sourceIndicesAndTargetLength);
WSPutList(Real32, target, sourceIndicesAndTargetLength);
WSReleaseList(source, source_length, Real32);
WSReleaseList(sourceIndices, sourceIndices_length, Integer32);
memoryFree(target);
}

__global__ void KERNEL_extract(real * const target, real const * const source, size_t const sourceLength, int const * const sourceIndices, size_t const sourceIndicesAndTargetLength)
{
extract(target, source, sourceLength, sourceIndices, sourceIndicesAndTargetLength);
}

extern "C" void extract_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(float, source, source_length, Real32);
WSGetList(int, sourceIndices, sourceIndices_length, Integer32);
float * source_in;
source_in = mallocmemcpy(source, source_length);
int * sourceIndices_in;
sourceIndices_in = mallocmemcpy(sourceIndices, sourceIndices_length);
int sourceLength;
sourceLength = source_length;
int sourceIndicesAndTargetLength;
sourceIndicesAndTargetLength = sourceIndices_length;
float * target;
target = tmalloc<float>(sourceIndicesAndTargetLength);
CUDAKERNEL_LAUNCH(KERNEL_extract, gridDim, blockDim, target, source_in, sourceLength, sourceIndices_in, sourceIndicesAndTargetLength);
WSPutList(Real32, target, sourceIndicesAndTargetLength);
WSReleaseList(source, source_length, Real32);
WSReleaseList(sourceIndices, sourceIndices_length, Integer32);
memoryFree(source_in);
memoryFree(sourceIndices_in);
memoryFree(target);
}

__host__ __device__ void writeFx(size_t const i, real const val);

extern "C" void writeFx_()
{
WSGet(long long, i, Integer64);
WSGet(float, val, Real32);
writeFx(i, val);
WL_RETURN_VOID();
}

__global__ void KERNEL_writeFx(size_t const i, real const val)
{
writeFx(i, val);
}

extern "C" void writeFx_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, i, Integer64);
WSGet(float, val, Real32);
CUDAKERNEL_LAUNCH(KERNEL_writeFx, gridDim, blockDim, i, val);
WL_RETURN_VOID();
}

__host__ __device__ void readZ(real * z, size_t const rowz);

extern "C" void readZ_()
{
WSGet(long long, rowz, Integer64);
float * z;
z = tmalloc<float>(lengthz);
readZ(z, rowz);
WSPutList(Real32, z, lengthz);
memoryFree(z);
}

__global__ void KERNEL_readZ(real * z, size_t const rowz)
{
readZ(z, rowz);
}

extern "C" void readZ_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, rowz, Integer64);
float * z;
z = tmalloc<float>(lengthz);
CUDAKERNEL_LAUNCH(KERNEL_readZ, gridDim, blockDim, z, rowz);
WSPutList(Real32, z, lengthz);
memoryFree(z);
}

__host__ __device__ void readZandSetFxRow(real * z, size_t const rowz, size_t const rowfz);

extern "C" void readZandSetFxRow_()
{
WSGet(long long, rowz, Integer64);
WSGet(long long, rowfz, Integer64);
float * z;
z = tmalloc<float>(lengthz);
readZandSetFxRow(z, rowz, rowfz);
WSPutList(Real32, z, lengthz);
memoryFree(z);
}

__global__ void KERNEL_readZandSetFxRow(real * z, size_t const rowz, size_t const rowfz)
{
readZandSetFxRow(z, rowz, rowfz);
}

extern "C" void readZandSetFxRow_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, rowz, Integer64);
WSGet(long long, rowfz, Integer64);
float * z;
z = tmalloc<float>(lengthz);
CUDAKERNEL_LAUNCH(KERNEL_readZandSetFxRow, gridDim, blockDim, z, rowz, rowfz);
WSPutList(Real32, z, lengthz);
memoryFree(z);
}

__host__ __device__ void setFxRow(size_t const rowz, size_t const rowfz);

extern "C" void setFxRow_()
{
WSGet(long long, rowz, Integer64);
WSGet(long long, rowfz, Integer64);
setFxRow(rowz, rowfz);
WL_RETURN_VOID();
}

__global__ void KERNEL_setFxRow(size_t const rowz, size_t const rowfz)
{
setFxRow(rowz, rowfz);
}

extern "C" void setFxRow_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, rowz, Integer64);
WSGet(long long, rowfz, Integer64);
CUDAKERNEL_LAUNCH(KERNEL_setFxRow, gridDim, blockDim, rowz, rowfz);
WL_RETURN_VOID();
}

__host__ __device__ void buildFx();

extern "C" void buildFx_()
{
buildFx();
WL_RETURN_VOID();
}

__global__ void KERNEL_buildFx()
{
buildFx();
}

extern "C" void buildFx_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_buildFx, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ real norm2Fx();

extern "C" void norm2Fx_()
{
real _result_;
_result_ = norm2Fx();
WSPut(Real32, _result_);
}

__global__ void KERNEL_norm2Fx(float * _returns_)
{
_returns_[linear_global_threadId()] = norm2Fx();
}

extern "C" void norm2Fx_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
float * _returns_;
_returns_ = tmalloc<float>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_norm2Fx, gridDim, blockDim, _returns_);
WSPutList(Real32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ float addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore();

extern "C" void addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore_()
{
float _result_;
_result_ = addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore();
WSPut(Real32, _result_);
}

__global__ void KERNEL_addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore(float * _returns_)
{
_returns_[linear_global_threadId()] = addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore();
}

extern "C" void addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
float * _returns_;
_returns_ = tmalloc<float>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_addContinuouslySmallerMultiplesOfHtoXUntilNorm2FxIsSmallerThanBefore, gridDim, blockDim, _returns_);
WSPutList(Real32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ void getY(real * const outY, int lengthY);

extern "C" void getY_()
{
WSGet(int, lengthY, Integer32);
float * outY;
outY = tmalloc<float>(lengthY);
getY(outY, lengthY);
WSPutList(Real32, outY, lengthY);
memoryFree(outY);
}

__global__ void KERNEL_getY(real * const outY, int lengthY)
{
getY(outY, lengthY);
}

extern "C" void getY_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, lengthY, Integer32);
float * outY;
outY = tmalloc<float>(lengthY);
CUDAKERNEL_LAUNCH(KERNEL_getY, gridDim, blockDim, outY, lengthY);
WSPutList(Real32, outY, lengthY);
memoryFree(outY);
}

__host__ __device__ void buildFxAndJFxAndSolveRepeatedly(int const iterations);

extern "C" void buildFxAndJFxAndSolveRepeatedly_()
{
WSGet(int, iterations, Integer32);
buildFxAndJFxAndSolveRepeatedly(iterations);
WL_RETURN_VOID();
}

__global__ void KERNEL_buildFxAndJFxAndSolveRepeatedly(int const iterations)
{
buildFxAndJFxAndSolveRepeatedly(iterations);
}

extern "C" void buildFxAndJFxAndSolveRepeatedly_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, iterations, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_buildFxAndJFxAndSolveRepeatedly, gridDim, blockDim, iterations);
WL_RETURN_VOID();
}

__host__ __device__ void receiveAndPrintOptimizationData(int const lengthz, int const lengthfz, real const * const x, size_t const xLength, int const * const sparseDerivativeZtoYIndices, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndices, size_t const xIndicesLength, int const * const yIndices, size_t const yIndicesLength);

extern "C" void receiveAndPrintOptimizationData_()
{
WSGet(int, lengthz, Integer32);
WSGet(int, lengthfz, Integer32);
WSGetList(float, x, x_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSGetList(int, xIndices, xIndices_length, Integer32);
WSGetList(int, yIndices, yIndices_length, Integer32);
int xLength;
xLength = x_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndices_length;
int xIndicesLength;
xIndicesLength = xIndices_length;
int yIndicesLength;
yIndicesLength = yIndices_length;
receiveAndPrintOptimizationData(lengthz, lengthfz, x, xLength, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndicesLength, xIndices, xIndicesLength, yIndices, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSReleaseList(xIndices, xIndices_length, Integer32);
WSReleaseList(yIndices, yIndices_length, Integer32);
}

__global__ void KERNEL_receiveAndPrintOptimizationData(int const lengthz, int const lengthfz, real const * const x, size_t const xLength, int const * const sparseDerivativeZtoYIndices, size_t const sparseDerivativeZtoYIndicesLength, int const * const xIndices, size_t const xIndicesLength, int const * const yIndices, size_t const yIndicesLength)
{
receiveAndPrintOptimizationData(lengthz, lengthfz, x, xLength, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndicesLength, xIndices, xIndicesLength, yIndices, yIndicesLength);
}

extern "C" void receiveAndPrintOptimizationData_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, lengthz, Integer32);
WSGet(int, lengthfz, Integer32);
WSGetList(float, x, x_length, Real32);
WSGetList(int, sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSGetList(int, xIndices, xIndices_length, Integer32);
WSGetList(int, yIndices, yIndices_length, Integer32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int * sparseDerivativeZtoYIndices_in;
sparseDerivativeZtoYIndices_in = mallocmemcpy(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length);
int * xIndices_in;
xIndices_in = mallocmemcpy(xIndices, xIndices_length);
int * yIndices_in;
yIndices_in = mallocmemcpy(yIndices, yIndices_length);
int xLength;
xLength = x_length;
int sparseDerivativeZtoYIndicesLength;
sparseDerivativeZtoYIndicesLength = sparseDerivativeZtoYIndices_length;
int xIndicesLength;
xIndicesLength = xIndices_length;
int yIndicesLength;
yIndicesLength = yIndices_length;
CUDAKERNEL_LAUNCH(KERNEL_receiveAndPrintOptimizationData, gridDim, blockDim, lengthz, lengthfz, x_in, xLength, sparseDerivativeZtoYIndices_in, sparseDerivativeZtoYIndicesLength, xIndices_in, xIndicesLength, yIndices_in, yIndicesLength);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(sparseDerivativeZtoYIndices, sparseDerivativeZtoYIndices_length, Integer32);
WSReleaseList(xIndices, xIndices_length, Integer32);
WSReleaseList(yIndices, yIndices_length, Integer32);
memoryFree(x_in);
memoryFree(sparseDerivativeZtoYIndices_in);
memoryFree(xIndices_in);
memoryFree(yIndices_in);
}

__host__ __device__ void makeAndPrintSparseMatrix(size_t const m, size_t const n, real * x, size_t xlen, int * ij, size_t const ijlen);

extern "C" void makeAndPrintSparseMatrix_()
{
WSGet(long long, m, Integer64);
WSGet(long long, n, Integer64);
WSGetList(float, x, x_length, Real32);
WSGetList(int, ij, ij_length, Integer32);
int xlen;
xlen = x_length;
int ijlen;
ijlen = ij_length;
makeAndPrintSparseMatrix(m, n, x, xlen, ij, ijlen);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(ij, ij_length, Integer32);
}

__global__ void KERNEL_makeAndPrintSparseMatrix(size_t const m, size_t const n, real * x, size_t xlen, int * ij, size_t const ijlen)
{
makeAndPrintSparseMatrix(m, n, x, xlen, ij, ijlen);
}

extern "C" void makeAndPrintSparseMatrix_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(long long, m, Integer64);
WSGet(long long, n, Integer64);
WSGetList(float, x, x_length, Real32);
WSGetList(int, ij, ij_length, Integer32);
float * x_in;
x_in = mallocmemcpy(x, x_length);
int * ij_in;
ij_in = mallocmemcpy(ij, ij_length);
int xlen;
xlen = x_length;
int ijlen;
ijlen = ij_length;
CUDAKERNEL_LAUNCH(KERNEL_makeAndPrintSparseMatrix, gridDim, blockDim, m, n, x_in, xlen, ij_in, ijlen);
WL_RETURN_VOID();
WSReleaseList(x, x_length, Real32);
WSReleaseList(ij, ij_length, Integer32);
memoryFree(x_in);
memoryFree(ij_in);
}

__host__ __device__ void testMain();

extern "C" void testMain_()
{
testMain();
WL_RETURN_VOID();
}

__global__ void KERNEL_testMain()
{
testMain();
}

extern "C" void testMain_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_testMain, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ void xx43();

extern "C" void xx43_()
{
xx43();
WL_RETURN_VOID();
}

__global__ void KERNEL_xx43()
{
xx43();
}

extern "C" void xx43_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
CUDAKERNEL_LAUNCH(KERNEL_xx43, gridDim, blockDim);
WL_RETURN_VOID();
}

__host__ __device__ int get42();

extern "C" void get42_()
{
int _result_;
_result_ = get42();
WSPut(Integer32, _result_);
}

__global__ void KERNEL_get42(int * _returns_)
{
_returns_[linear_global_threadId()] = get42();
}

extern "C" void get42_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_get42, gridDim, blockDim, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int add(int x, int y);

extern "C" void add_()
{
WSGet(int, x, Integer32);
WSGet(int, y, Integer32);
int _result_;
_result_ = add(x, y);
WSPut(Integer32, _result_);
}

__global__ void KERNEL_add(int x, int y, int * _returns_)
{
_returns_[linear_global_threadId()] = add(x, y);
}

extern "C" void add_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(int, x, Integer32);
WSGet(int, y, Integer32);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_add, gridDim, blockDim, x, y, _returns_);
WSPutList(Integer32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ real addf(real x, real y);

extern "C" void addf_()
{
WSGet(float, x, Real32);
WSGet(float, y, Real32);
real _result_;
_result_ = addf(x, y);
WSPut(Real32, _result_);
}

__global__ void KERNEL_addf(real x, real y, float * _returns_)
{
_returns_[linear_global_threadId()] = addf(x, y);
}

extern "C" void addf_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGet(float, x, Real32);
WSGet(float, y, Real32);
float * _returns_;
_returns_ = tmalloc<float>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_addf, gridDim, blockDim, x, y, _returns_);
WSPutList(Real32, _returns_, gridDim * blockDim);
memoryFree(_returns_);
}

__host__ __device__ int multiout(int * x, int l);

extern "C" void multiout_()
{
WSGetList(int, x, x_length, Integer32);
int l;
l = x_length;
int * x_inout;
x_inout = mallocmemcpy(x, l);
int _result_;
_result_ = multiout(x_inout, l);
WSPutFunction(stdlink, "Association", 2);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("return");
WSPut(Integer32, _result_);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("x");
WSPutList(Integer32, x_inout, l);
WSReleaseList(x, x_length, Integer32);
memoryFree(x_inout);
}

__global__ void KERNEL_multiout(int * x_inout, int l, int * _returns_)
{
_returns_[linear_global_threadId()] = multiout(x_inout, l);
}

extern "C" void multiout_CUDA()
{
WSGet(int, gridDim, Integer32);
WSGet(int, blockDim, Integer32);
WSGetList(int, x, x_length, Integer32);
int l;
l = x_length;
int * x_inout;
x_inout = mallocmemcpy(x, l);
int * _returns_;
_returns_ = tmalloc<int>(gridDim * blockDim);
CUDAKERNEL_LAUNCH(KERNEL_multiout, gridDim, blockDim, x_inout, l, _returns_);
WSPutFunction(stdlink, "Association", 2);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("x");
WSPutList(Integer32, x_inout, l);
WSPutFunction(stdlink, "Rule", 2);
WSPutCString("returns");
WSPutList(Integer32, _returns_, gridDim * blockDim);
WSReleaseList(x, x_length, Integer32);
memoryFree(x_inout);
memoryFree(_returns_);
}

extern int dprintEnabled;

extern "C" void dprintEnabled_get()
{
WSPut(Integer32, dprintEnabled);
}

extern int lengthP;

extern "C" void lengthP_get()
{
WSPut(Integer32, lengthP);
}

extern int xx;

extern "C" void xx_get()
{
WSPut(Integer32, xx);
}

extern real a;

extern "C" void a_get()
{
WSPut(Real32, a);
}

extern real b;

extern "C" void b_get()
{
WSPut(Real32, b);
}

extern real yy;

extern "C" void yy_get()
{
WSPut(Real32, yy);
}

extern real * x;

extern int lengthx;

extern "C" void x_get()
{
WSPutList(Real32, x, lengthx);
}

extern real * minusFx;

extern int lengthFx;

extern "C" void minusFx_get()
{
WSPutList(Real32, minusFx, lengthFx);
}

extern real * h;

extern int lengthY;

extern "C" void h_get()
{
WSPutList(Real32, h, lengthY);
}

extern int * yIndices;

extern int lengthY;

extern "C" void yIndices_get()
{
WSPutList(Integer32, yIndices, lengthY);
}

extern "C" void dprintEnabled_set()
{
WSGetInteger32(stdlink,&dprintEnabled);
WL_RETURN_VOID();
}

extern "C" void lengthP_set()
{
WSGetInteger32(stdlink,&lengthP);
WL_RETURN_VOID();
}

extern "C" void xx_set()
{
WSGetInteger32(stdlink,&xx);
WL_RETURN_VOID();
}

extern "C" void a_set()
{
WSGetReal32(stdlink,&a);
WL_RETURN_VOID();
}

extern "C" void b_set()
{
WSGetReal32(stdlink,&b);
WL_RETURN_VOID();
}

extern "C" void yy_set()
{
WSGetReal32(stdlink,&yy);
WL_RETURN_VOID();
}

extern "C" void x_set()
{
WSGetList(float, x_in, x_in_length, Real32);
freemalloctmemcpy(&x, x_in, x_in_length);
lengthx = x_in_length;
WL_RETURN_VOID();
WSReleaseList(x_in, x_in_length, Real32);
}

extern "C" void minusFx_set()
{
WSGetList(float, minusFx_in, minusFx_in_length, Real32);
freemalloctmemcpy(&minusFx, minusFx_in, minusFx_in_length);
lengthFx = minusFx_in_length;
WL_RETURN_VOID();
WSReleaseList(minusFx_in, minusFx_in_length, Real32);
}

extern "C" void h_set()
{
WSGetList(float, h_in, h_in_length, Real32);
freemalloctmemcpy(&h, h_in, h_in_length);
lengthY = h_in_length;
WL_RETURN_VOID();
WSReleaseList(h_in, h_in_length, Real32);
}

extern "C" void yIndices_set()
{
WSGetList(int, yIndices_in, yIndices_in_length, Integer32);
freemalloctmemcpy(&yIndices, yIndices_in, yIndices_in_length);
lengthY = yIndices_in_length;
WL_RETURN_VOID();
WSReleaseList(yIndices_in, yIndices_in_length, Integer32);
}

